#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1)
{
    return x0 + x1;
}
__device__ __forceinline__ float relu(float x0)
{
    return fmaxf(0,x0);
}
// Node name:	Constant_301
// Description:	Constant
// Input:
// Output:
//	- name: Constant_301_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_301(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_301_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_301_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4608];
    bin_file.read(tmp_mem, 4608);
    hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_285
// Description:	Constant
// Input:
// Output:
//	- name: Constant_285_0	type: float	shape: Shape{5, 5, 64, 1}
void Constant_float_cuda_Constant_285(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_285_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_285_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[6400];
    bin_file.read(tmp_mem, 6400);
    hipMemcpyAsync(output0, tmp_mem, 6400, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_7
// Description:	Constant
// Input:
// Output:
//	- name: Constant_7_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_7(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_7_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_7_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[2304];
    bin_file.read(tmp_mem, 2304);
    hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2781
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2781_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2781(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2781_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2781_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[131072];
    bin_file.read(tmp_mem, 131072);
    hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2722
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2722_0	type: float	shape: Shape{128, 768, 1, 1}
void Constant_float_cuda_Constant_2722(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2722_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2722_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[393216];
    bin_file.read(tmp_mem, 393216);
    hipMemcpyAsync(output0, tmp_mem, 393216, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2911
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2911_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2911(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2911_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2911_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2500
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2500_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2500(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2500_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2500_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_1338
// Description:	Constant
// Input:
// Output:
//	- name: Constant_1338_0	type: float	shape: Shape{}
void Constant_float_cuda_Constant_1338(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_1338_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_1338_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4];
    bin_file.read(tmp_mem, 4);
    hipMemcpyAsync(output0, tmp_mem, 4, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_458
// Description:	Constant
// Input:
// Output:
//	- name: Constant_458_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_458(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_458_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_458_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4608];
    bin_file.read(tmp_mem, 4608);
    hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_35
// Description:	Constant
// Input:
// Output:
//	- name: Constant_35_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_35(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_35_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_35_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4608];
    bin_file.read(tmp_mem, 4608);
    hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_3030
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3030_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_3030(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_3030_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_3030_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[131072];
    bin_file.read(tmp_mem, 131072);
    hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_871_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2254_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: DepthwiseConv2dNative_872_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2257_0	type: float	shape: Shape{32, 32, 1, 1}
// Output:
//	- name: Convolution_877_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_879_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_877<<<dim3(2, 16, 2), dim3(16, 2, 8), 0, 0>>>(DepthwiseConv2dNative_871_0, Constant_2254_0, Convolution_877_0);
// Convolution_float_float_float_cuda_Convolution_879<<<dim3(2, 16, 2), dim3(16, 2, 8), 0, 0>>>(DepthwiseConv2dNative_872_0, Constant_2257_0, Convolution_879_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Convolution_float_float_float_cuda_Convolution_879 : Convolution_float_float_float_cuda_Convolution_877

// Node name:	Convolution_877
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_871_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2254_0	type: float	shape: Shape{32, 32, 1, 1}
// Output:
//	- name: Convolution_877_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_877_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(16, 2, 8);
    const dim3 gridDim(2, 16, 2);
    const dim3 threadIdx(thread_id % 16, thread_id / 16 % 2, thread_id / 32);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 16, block_id / 32);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 2048);
    {
        float* compute = output0;{
           float compute_local[2];
          
          
          for (int ff_c_init = 0; ff_c_init < 2; ++ff_c_init) {
            compute_local[ff_c_init] = 0.000000e+00f;
          }
          for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
            __syncthreads();
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
              pad_temp_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] = input0[(((((((rc_outer * 16384) + (((int)threadIdx.z) * 2048)) + (((int)threadIdx.y) * 1024)) + (((int)blockIdx.y) * 64)) + ((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 4) * 32)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 15))];
            }
            input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 32)) + (rc_outer * 16)) + ((int)threadIdx.x))];
            __syncthreads();
            for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
              for (int ff_c = 0; ff_c < 2; ++ff_c) {
                compute_local[ff_c] = (compute_local[ff_c] + (pad_temp_shared[(((rc_inner * 32) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x))] * input1_shared[(((((int)threadIdx.z) * 32) + (ff_c * 16)) + rc_inner)]));
              }
            }
          }
          for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 2; ++ff_inner_inner_inner) {
            compute[(((((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 2048)) + (ff_inner_inner_inner * 1024)) + (((int)blockIdx.y) * 64)) + (((int)threadIdx.y) * 32)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x))] = compute_local[ff_inner_inner_inner];
          }
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_52(float* input0, float* input1, float* input2, float* input3, float* output0, float* output1)
{
    __shared__ char shared_buffer[3072];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Convolution_float_float_float_cuda_Convolution_877_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Convolution_float_float_float_cuda_Convolution_877_block_kernel(input2, input3, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_52_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_52<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0, output1);
}
// Node name:	Convolution_466
// Description:	Convolution
// Input:
//	- name: Reshape_464_0	type: float	shape: Shape{1, 3, 32, 32}
//	- name: Constant_2011_0	type: float	shape: Shape{96, 3, 3, 3}
// Output:
//	- name: Convolution_466_0	type: float	shape: Shape{1, 96, 32, 32}
extern "C" __global__  void Convolution_float_float_float_cuda_Convolution_466(float* input0, float* input1, float* output0)
{
    __shared__ float pad_temp_shared[408];
    __shared__ float input1_shared[864];
    {
        float* compute = output0;{
           float compute_local[4];
          
          
          #pragma unroll
          for (int yy_c_init = 0; yy_c_init < 2; ++yy_c_init) {
            compute_local[yy_c_init] = 0.000000e+00f;
            compute_local[(yy_c_init + 2)] = 0.000000e+00f;
          }
          if (((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) < 408) {
            if (((int)threadIdx.x) < 26) {
              pad_temp_shared[((((int)threadIdx.z) * 26) + ((int)threadIdx.x))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) % 136) / 34))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) % 136) / 34)) < 33)) && (1 <= (((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) % 34))) && ((((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) % 34) < 33)) ? input0[((((((((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) / 136) * 1024) + (((int)blockIdx.y) * 64)) + (((((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) % 136) / 34) * 32)) + (((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) % 34)) - 33)] : 0.000000e+00f);
            }
          }
          #pragma unroll
          for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
            if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 27)) < 32) {
              if (((((int)threadIdx.z) * 6) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 9)) < 96) {
                if (((((int)threadIdx.z) * 18) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 3)) < 288) {
                  if ((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 864) {
                    if (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 54) {
                      input1_shared[(((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] = input1[((((((int)blockIdx.z) * 864) + (((int)threadIdx.z) * 54)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)];
                    }
                  }
                }
              }
            }
          }
          __syncthreads();
          #pragma unroll
          for (int rc_inner = 0; rc_inner < 3; ++rc_inner) {
            #pragma unroll
            for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
              #pragma unroll
              for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
                #pragma unroll
                for (int yy_c = 0; yy_c < 2; ++yy_c) {
                  compute_local[yy_c] = (compute_local[yy_c] + (pad_temp_shared[(((((rc_inner * 136) + (yy_c * 34)) + (ry_inner * 34)) + ((int)threadIdx.x)) + rx_inner)] * input1_shared[((((((int)threadIdx.z) * 27) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
                  compute_local[(yy_c + 2)] = (compute_local[(yy_c + 2)] + (pad_temp_shared[(((((rc_inner * 136) + (yy_c * 34)) + (ry_inner * 34)) + ((int)threadIdx.x)) + rx_inner)] * input1_shared[(((((((int)threadIdx.z) * 27) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 432)]));
                }
              }
            }
          }
          #pragma unroll
          for (int yy_inner_inner_inner = 0; yy_inner_inner_inner < 2; ++yy_inner_inner_inner) {
            compute[(((((((int)blockIdx.z) * 32768) + (((int)threadIdx.z) * 1024)) + (((int)blockIdx.y) * 64)) + (yy_inner_inner_inner * 32)) + ((int)threadIdx.x))] = compute_local[yy_inner_inner_inner];
            compute[((((((((int)blockIdx.z) * 32768) + (((int)threadIdx.z) * 1024)) + (((int)blockIdx.y) * 64)) + (yy_inner_inner_inner * 32)) + ((int)threadIdx.x)) + 16384)] = compute_local[(yy_inner_inner_inner + 2)];
          }
        }


    }

}
extern void Convolution_float_float_float_cuda_Convolution_466_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0) {
    Convolution_float_float_float_cuda_Convolution_466<<<grids, blocks, mem, stream>>>(input0, input1, output0);
}
// Node name:	Constant_2741
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2741_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2741(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2741_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2741_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[131072];
    bin_file.read(tmp_mem, 131072);
    hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	 BlockFusion
// Input:
//	- name: Slice_1094_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: AvgPool_1096_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1101_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2377_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3082_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1099_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2371_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3078_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1100_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2374_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3080_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_1097_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Add_1102_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_1123_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_1121_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_1122_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Relu_float_float_cuda_Relu_1097<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(Slice_1094_0, Relu_1097_0);
// Add_float_float_float_cuda_Add_1102<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(AvgPool_1096_0, AvgPool_1096_0, Add_1102_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3081<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1101_0, Constant_2377_0, Constant_3082_0, Relu_1123_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3077<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1099_0, Constant_2371_0, Constant_3078_0, Relu_1121_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3079<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1100_0, Constant_2374_0, Constant_3080_0, Relu_1122_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3077 : Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3081
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3079 : Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3081

// Node name:	Relu_1097
// Description:	Relu
// Input:
//	- name: Slice_1094_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_1097_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Relu_float_float_cuda_Relu_1097_block_kernel(float* input0, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(32, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = relu(input0[blockIdx.x * 512 + threadIdx.x]);

}
// Node name:	Add_1102
// Description:	Add
// Input:
//	- name: AvgPool_1096_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: AvgPool_1096_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Add_1102_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Add_float_float_float_cuda_Add_1102_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(32, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = add(input0[blockIdx.x * 512 + threadIdx.x], input1[blockIdx.x * 512 + threadIdx.x]);

}
// Node name:	Matched_Pattern_3081
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_1101_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2377_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3082_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_1123_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3081_block_kernel(float* input0, float* input1, float* input2, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(8, 2, 16);
    const dim3 gridDim(2, 8, 4);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 8, block_id / 16);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 1024);
    {
        float* compute = output0;{
           float compute1[1];
          
          
          compute1[0] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[(((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 4096)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 16)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 8192)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 32)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 12288)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 48)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          compute[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))] = max((compute1[0] + input2[((((int)blockIdx.z) * 16) + ((int)threadIdx.z))]), 0.000000e+00f);
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Add_Matched_Pattern_Matched_Pattern_Matched_Pattern_86(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* output0, float* output1, float* output2, float* output3, float* output4)
{
    __shared__ char shared_buffer[2048];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 31)
    {
        Relu_float_float_cuda_Relu_1097_block_kernel(input0, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 63)
    {
        Add_float_float_float_cuda_Add_1102_block_kernel(input1, input1, output1, threadIdx.x, blockIdx.x - 32 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3081_block_kernel(input2, input3, input4, output2, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3081_block_kernel(input5, input6, input7, output3, threadIdx.x, blockIdx.x - 128 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 192 && (int)blockIdx.x <= 255)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3081_block_kernel(input8, input9, input10, output4, threadIdx.x, blockIdx.x - 192 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Add_Matched_Pattern_Matched_Pattern_Matched_Pattern_86_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* output0, float* output1, float* output2, float* output3, float* output4) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Add_Matched_Pattern_Matched_Pattern_Matched_Pattern_86<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, input6, input7, input8, input9, input10, output0, output1, output2, output3, output4);
}
