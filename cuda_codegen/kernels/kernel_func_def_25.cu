#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1)
{
    return x0 + x1;
}
__device__ __forceinline__ float relu(float x0)
{
    return fmaxf(0,x0);
}
// Node name:	Constant_441
// Description:	Constant
// Input:
// Output:
//	- name: Constant_441_0	type: float	shape: Shape{5, 5, 128, 1}
void Constant_float_cuda_Constant_441(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_441_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_441_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[12800];
    bin_file.read(tmp_mem, 12800);
    hipMemcpyAsync(output0, tmp_mem, 12800, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2641
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2641_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2641(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2641_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2641_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2044
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2044_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2044(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2044_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2044_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4096];
    bin_file.read(tmp_mem, 4096);
    hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2617
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2617_0	type: float	shape: Shape{128, 768, 1, 1}
void Constant_float_cuda_Constant_2617(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2617_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2617_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[393216];
    bin_file.read(tmp_mem, 393216);
    hipMemcpyAsync(output0, tmp_mem, 393216, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2470
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2470_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2470(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2470_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2470_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2900
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2900_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2900(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2900_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2900_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2251
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2251_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2251(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2251_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2251_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_297
// Description:	Constant
// Input:
// Output:
//	- name: Constant_297_0	type: float	shape: Shape{5, 5, 64, 1}
void Constant_float_cuda_Constant_297(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_297_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_297_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[6400];
    bin_file.read(tmp_mem, 6400);
    hipMemcpyAsync(output0, tmp_mem, 6400, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2967
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2967_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2967(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2967_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2967_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2843
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2843_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2843(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2843_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2843_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[131072];
    bin_file.read(tmp_mem, 131072);
    hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	 BlockFusion
// Input:
//	- name: Convolution_701_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2746_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_703_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2847_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: BatchNormInference_704_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Relu_707_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: BatchNormInference_705_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// FusedKernel_float_float_float_float_cuda_Add_Relu_15<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_701_0, Constant_2746_0, Relu_707_0, BatchNormInference_704_0);
// Add_float_float_float_cuda_Add_2157<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_703_0, Constant_2847_0, BatchNormInference_705_0);
// Deduped function map: <src_function_name : deduped_function_name>

// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_701_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2746_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Relu_707_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: BatchNormInference_704_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Add_float_float_float_cuda_Add_2154<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_701_0, Constant_2746_0, BatchNormInference_704_0);
// Relu_float_float_cuda_Relu_707<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(BatchNormInference_704_0, Relu_707_0);
__device__ __noinline__ void FusedKernel_float_float_float_float_cuda_Add_Relu_15_block_kernel(float* input0, float* input1, float* output0, float* output1, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    int tid = blockIdx.x * 512 + threadIdx.x;
    float temp0 = add(input0[tid], input1[tid]);
    float temp1 = relu(temp0);
    output1[tid] = temp0;
    output0[tid] = temp1;

}
// Node name:	Add_2157
// Description:	Add
// Input:
//	- name: Convolution_703_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2847_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: BatchNormInference_705_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void Add_float_float_float_cuda_Add_2157_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = add(input0[blockIdx.x * 512 + threadIdx.x], input1[blockIdx.x * 512 + threadIdx.x]);

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_cuda_fused_kernel_Add_28(float* input0, float* input1, float* input2, float* input3, float* output0, float* output1, float* output2)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        FusedKernel_float_float_float_float_cuda_Add_Relu_15_block_kernel(input0, input1, output1, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Add_float_float_float_cuda_Add_2157_block_kernel(input2, input3, output2, threadIdx.x, blockIdx.x - 64 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_cuda_fused_kernel_Add_28_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* output0, float* output1, float* output2) {
    BlockFusionKernel_float_float_float_float_float_float_float_cuda_fused_kernel_Add_28<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0, output1, output2);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_904_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2263_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3048_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_76_0	type: float	shape: Shape{64}
//	- name: Constant_55_0	type: float	shape: Shape{64}
//	- name: Concat_905_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_912_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: BatchNormInference_908_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3047<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_904_0, Constant_2263_0, Constant_3048_0, Relu_912_0);
// BatchNormInference_float_float_float_float_float_float_cuda_BatchNormInference_908<<<dim3(64, 1, 1), dim3(256, 1, 1), 0, 0>>>(Constant_76_0, Constant_55_0, Concat_905_0, Constant_55_0, Constant_76_0, BatchNormInference_908_0);
// Deduped function map: <src_function_name : deduped_function_name>

// Node name:	Matched_Pattern_3047
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_904_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2263_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3048_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_912_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3047_block_kernel(float* input0, float* input1, float* input2, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(8, 2, 16);
    const dim3 gridDim(2, 8, 4);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 8, block_id / 16);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 1024);
    {
        float* compute = output0;{
           float compute1[1];
          
          
          compute1[0] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[(((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 4096)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 16)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 8192)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 32)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 12288)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 48)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          compute[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))] = max((compute1[0] + input2[((((int)blockIdx.z) * 16) + ((int)threadIdx.z))]), 0.000000e+00f);
        }


    }

}
// Node name:	BatchNormInference_908
// Description:	BatchNormInference
// Input:
//	- name: Constant_76_0	type: float	shape: Shape{64}
//	- name: Constant_55_0	type: float	shape: Shape{64}
//	- name: Concat_905_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_55_0	type: float	shape: Shape{64}
//	- name: Constant_76_0	type: float	shape: Shape{64}
// Output:
//	- name: BatchNormInference_908_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void BatchNormInference_float_float_float_float_float_float_cuda_BatchNormInference_908_block_kernel(float* input0, float* input1, float* input2, float* input3, float* input4, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(256, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    const int st = blockIdx.x * 16 * 16;
    const int c_id = blockIdx.x % 64;
    #pragma unroll 1
    for (int i = threadIdx.x; i < 16 * 16; i += blockDim.x)
    {
        output0[st + i] = (input1[c_id] + (input0[c_id] * (input2[st + i] - input3[c_id]) / sqrtf(1.001e-05 + input4[c_id])));
    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_BatchNormInference_58(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* output0, float* output1)
{
    __shared__ char shared_buffer[2048];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3047_block_kernel(input0, input1, input2, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        BatchNormInference_float_float_float_float_float_float_cuda_BatchNormInference_908_block_kernel(input3, input4, input5, input4, input3, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_BatchNormInference_58_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_BatchNormInference_58<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, output0, output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_1555_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2629_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3156_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1556_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2632_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3158_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1577_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2635_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: DepthwiseConv2dNative_1579_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2641_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: DepthwiseConv2dNative_1578_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2638_0	type: float	shape: Shape{128, 128, 1, 1}
// Output:
//	- name: Relu_1575_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1576_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1583_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1587_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1585_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3155<<<dim3(1, 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1555_0, Constant_2629_0, Constant_3156_0, Relu_1575_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3157<<<dim3(1, 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1556_0, Constant_2632_0, Constant_3158_0, Relu_1576_0);
// Convolution_float_float_float_cuda_Convolution_1583<<<dim3(1, 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1577_0, Constant_2635_0, Convolution_1583_0);
// Convolution_float_float_float_cuda_Convolution_1587<<<dim3(1, 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1579_0, Constant_2641_0, Convolution_1587_0);
// Convolution_float_float_float_cuda_Convolution_1585<<<dim3(1, 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1578_0, Constant_2638_0, Convolution_1585_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3157 : Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3155
// Convolution_float_float_float_cuda_Convolution_1587 : Convolution_float_float_float_cuda_Convolution_1583
// Convolution_float_float_float_cuda_Convolution_1585 : Convolution_float_float_float_cuda_Convolution_1583

// Node name:	Matched_Pattern_3155
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_1555_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2629_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3156_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1575_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3155_block_kernel(float* input0, float* input1, float* input2, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(8, 2, 8);
    const dim3 gridDim(1, 4, 16);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 1, block_id / 1 % 4, block_id / 4);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 1024);
    {
        float* compute = output0;{
           float compute1[1];
          
          
          compute1[0] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2))];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1024)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1025)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 16)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2048)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2049)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 32)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 3072)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 3073)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 48)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 4096)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 4097)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 64)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 5120)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 5121)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 80)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 6144)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 6145)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 96)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 7168)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 7169)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 112)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          compute[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = max((compute1[0] + input2[((((int)blockIdx.z) * 8) + ((int)threadIdx.z))]), 0.000000e+00f);
        }


    }

}
// Node name:	Convolution_1583
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_1577_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2635_0	type: float	shape: Shape{128, 128, 1, 1}
// Output:
//	- name: Convolution_1583_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_1583_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(8, 2, 8);
    const dim3 gridDim(1, 4, 16);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 1, block_id / 1 % 4, block_id / 4);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 1024);
    {
        float* compute = output0;{
           float compute_local[1];
          
          
          compute_local[0] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2))];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1024)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1025)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 16)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2048)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2049)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 32)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 3072)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 3073)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 48)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 4096)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 4097)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 64)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 5120)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 5121)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 80)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 6144)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 6145)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 96)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 7168)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 7169)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 112)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          compute[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = compute_local[0];
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Convolution_Convolution_Convolution_153(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* input11, float* output0, float* output1, float* output2, float* output3, float* output4)
{
    __shared__ char shared_buffer[1536];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3155_block_kernel(input0, input1, input2, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3155_block_kernel(input3, input4, input5, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191)
    {
        Convolution_float_float_float_cuda_Convolution_1583_block_kernel(input6, input7, output2, threadIdx.x, blockIdx.x - 128 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 192 && (int)blockIdx.x <= 255)
    {
        Convolution_float_float_float_cuda_Convolution_1583_block_kernel(input8, input9, output3, threadIdx.x, blockIdx.x - 192 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 256 && (int)blockIdx.x <= 319)
    {
        Convolution_float_float_float_cuda_Convolution_1583_block_kernel(input10, input11, output4, threadIdx.x, blockIdx.x - 256 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Convolution_Convolution_Convolution_153_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* input11, float* output0, float* output1, float* output2, float* output3, float* output4) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Convolution_Convolution_Convolution_153<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, input6, input7, input8, input9, input10, input11, output0, output1, output2, output3, output4);
}
// Node name:	 BlockFusion
// Input:
//	- name: AvgPool_1337_0	type: float	shape: Shape{1, 384, 8, 8}
//	- name: Constant_1934_0	type: float	shape: Shape{64, 384, 1, 1}
//	- name: Pad_1339_0	type: float	shape: Shape{1, 384, 17, 17}
//	- name: Convolution_1341_0	type: float	shape: Shape{1, 128, 16, 16}
//	- name: Constant_2764_0	type: float	shape: Shape{1, 128, 16, 16}
// Output:
//	- name: Convolution_1343_0	type: float	shape: Shape{1, 64, 8, 8}
//	- name: Slice_1344_0	type: float	shape: Shape{1, 384, 16, 16}
//	- name: BatchNormInference_1345_0	type: float	shape: Shape{1, 128, 16, 16}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_1343<<<dim3(1, 8, 8), dim3(8, 1, 8), 0, 0>>>(AvgPool_1337_0, Constant_1934_0, Convolution_1343_0);
// Slice_float_float_cuda_Slice_1344<<<dim3(1536, 1, 1), dim3(64, 1, 1), 0, 0>>>(Pad_1339_0, Slice_1344_0);
// Add_float_float_float_cuda_Add_2511<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1341_0, Constant_2764_0, BatchNormInference_1345_0);
// Deduped function map: <src_function_name : deduped_function_name>

// Node name:	Convolution_1343
// Description:	Convolution
// Input:
//	- name: AvgPool_1337_0	type: float	shape: Shape{1, 384, 8, 8}
//	- name: Constant_1934_0	type: float	shape: Shape{64, 384, 1, 1}
// Output:
//	- name: Convolution_1343_0	type: float	shape: Shape{1, 64, 8, 8}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_1343_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 64){
        return;
    }
    const dim3 blockDim(8, 1, 8);
    const dim3 gridDim(1, 8, 8);
    const dim3 threadIdx(thread_id % 8, 0, thread_id / 8);
    const dim3 blockIdx(block_id % 1, block_id / 1 % 8, block_id / 8);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 512);
    {
        float* compute = output0;{
           float compute_local[1];
          
          
          compute_local[0] = 0.000000e+00f;
          #pragma unroll
          for (int rc_outer = 0; rc_outer < 24; ++rc_outer) {
            __syncthreads();
            #pragma unroll
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
              pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] = input0[(((((rc_outer * 1024) + (((int)threadIdx.z) * 128)) + ((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 3) * 64)) + (((int)blockIdx.y) * 8)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 7))];
            }
            #pragma unroll
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
              input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)] = input1[(((((((int)blockIdx.z) * 3072) + (((int)threadIdx.z) * 384)) + (rc_outer * 16)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)];
            }
            __syncthreads();
            #pragma unroll
            for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
              compute_local[0] = (compute_local[0] + (pad_temp_shared[((rc_inner * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 16) + rc_inner)]));
            }
          }
          compute[((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) + (((int)blockIdx.y) * 8)) + ((int)threadIdx.x))] = compute_local[0];
        }


    }

}
// Node name:	Slice_1344
// Description:	Slice
// Input:
//	- name: Pad_1339_0	type: float	shape: Shape{1, 384, 17, 17}
// Output:
//	- name: Slice_1344_0	type: float	shape: Shape{1, 384, 16, 16}
__device__ __noinline__ void Slice_float_float_cuda_Slice_1344_block_kernel(float* input0, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 64){
        return;
    }
    const dim3 blockDim(64, 1, 1);
    const dim3 gridDim(1536, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < 98304)
    {
        uint32_t input_strides[] = {110976, 289, 17, 1};
        uint32_t output_strides[] = {98304, 256, 16, 1};
        uint32_t lower_bounds[] = {0, 0, 1, 1};
        uint32_t slice_strides[] = {1, 1, 1, 1};
        uint32_t input_idx = 0;
        uint32_t output_idx = tid;
        input_idx += (((output_idx / output_strides[0]) * slice_strides[0]) + lower_bounds[0]) * input_strides[0];
        output_idx %= output_strides[0];
        input_idx += (((output_idx / output_strides[1]) * slice_strides[1]) + lower_bounds[1]) * input_strides[1];
        output_idx %= output_strides[1];
        input_idx += (((output_idx / output_strides[2]) * slice_strides[2]) + lower_bounds[2]) * input_strides[2];
        output_idx %= output_strides[2];
        input_idx += (((output_idx / output_strides[3]) * slice_strides[3]) + lower_bounds[3]) * input_strides[3];
        output0[tid] = input0[input_idx];
    }

}
// Node name:	Add_2511
// Description:	Add
// Input:
//	- name: Convolution_1341_0	type: float	shape: Shape{1, 128, 16, 16}
//	- name: Constant_2764_0	type: float	shape: Shape{1, 128, 16, 16}
// Output:
//	- name: BatchNormInference_1345_0	type: float	shape: Shape{1, 128, 16, 16}
__device__ __noinline__ void Add_float_float_float_cuda_Add_2511_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = add(input0[blockIdx.x * 512 + threadIdx.x], input1[blockIdx.x * 512 + threadIdx.x]);

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_Convolution_Slice_Add_120(float* input0, float* input1, float* input2, float* input3, float* input4, float* output0, float* output1, float* output2)
{
    __shared__ char shared_buffer[1024];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Convolution_float_float_float_cuda_Convolution_1343_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 1599)
    {
        Slice_float_float_cuda_Slice_1344_block_kernel(input2, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 1600 && (int)blockIdx.x <= 1663)
    {
        Add_float_float_float_cuda_Add_2511_block_kernel(input3, input4, output2, threadIdx.x, blockIdx.x - 1600 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_Convolution_Slice_Add_120_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* output0, float* output1, float* output2) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_Convolution_Slice_Add_120<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, output0, output1, output2);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_1103_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2380_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3084_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1104_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2383_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3086_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1128_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2392_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: DepthwiseConv2dNative_1126_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2386_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: DepthwiseConv2dNative_1127_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2389_0	type: float	shape: Shape{64, 64, 1, 1}
// Output:
//	- name: Relu_1124_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_1125_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1136_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1132_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1134_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3083<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1103_0, Constant_2380_0, Constant_3084_0, Relu_1124_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3085<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1104_0, Constant_2383_0, Constant_3086_0, Relu_1125_0);
// Convolution_float_float_float_cuda_Convolution_1136<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1128_0, Constant_2392_0, Convolution_1136_0);
// Convolution_float_float_float_cuda_Convolution_1132<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1126_0, Constant_2386_0, Convolution_1132_0);
// Convolution_float_float_float_cuda_Convolution_1134<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1127_0, Constant_2389_0, Convolution_1134_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3085 : Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3083
// Convolution_float_float_float_cuda_Convolution_1132 : Convolution_float_float_float_cuda_Convolution_1136
// Convolution_float_float_float_cuda_Convolution_1134 : Convolution_float_float_float_cuda_Convolution_1136

// Node name:	Matched_Pattern_3083
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_1103_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2380_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3084_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_1124_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3083_block_kernel(float* input0, float* input1, float* input2, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(8, 2, 16);
    const dim3 gridDim(2, 8, 4);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 8, block_id / 16);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 1024);
    {
        float* compute = output0;{
           float compute1[1];
          
          
          compute1[0] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[(((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 4096)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 16)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 8192)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 32)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 12288)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 48)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          compute[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))] = max((compute1[0] + input2[((((int)blockIdx.z) * 16) + ((int)threadIdx.z))]), 0.000000e+00f);
        }


    }

}
// Node name:	Convolution_1136
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_1128_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2392_0	type: float	shape: Shape{64, 64, 1, 1}
// Output:
//	- name: Convolution_1136_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_1136_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(8, 2, 16);
    const dim3 gridDim(2, 8, 4);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 8, block_id / 16);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 1024);
    {
        float* compute = output0;{
           float compute_local[1];
          
          
          compute_local[0] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[(((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 4096)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 16)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 8192)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 32)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 12288)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 48)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          compute[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))] = compute_local[0];
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Convolution_Convolution_Convolution_88(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* input11, float* output0, float* output1, float* output2, float* output3, float* output4)
{
    __shared__ char shared_buffer[2048];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3083_block_kernel(input0, input1, input2, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3083_block_kernel(input3, input4, input5, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191)
    {
        Convolution_float_float_float_cuda_Convolution_1136_block_kernel(input6, input7, output2, threadIdx.x, blockIdx.x - 128 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 192 && (int)blockIdx.x <= 255)
    {
        Convolution_float_float_float_cuda_Convolution_1136_block_kernel(input8, input9, output3, threadIdx.x, blockIdx.x - 192 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 256 && (int)blockIdx.x <= 319)
    {
        Convolution_float_float_float_cuda_Convolution_1136_block_kernel(input10, input11, output4, threadIdx.x, blockIdx.x - 256 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Convolution_Convolution_Convolution_88_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* input11, float* output0, float* output1, float* output2, float* output3, float* output4) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Convolution_Convolution_Convolution_88<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, input6, input7, input8, input9, input10, input11, output0, output1, output2, output3, output4);
}
