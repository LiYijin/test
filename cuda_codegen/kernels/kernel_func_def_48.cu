#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1)
{
    return x0 + x1;
}
__device__ __forceinline__ float relu(float x0)
{
    return fmaxf(0,x0);
}
// Node name:	Constant_109
// Description:	Constant
// Input:
// Output:
//	- name: Constant_109_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_109(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_109_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_109_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4608];
    bin_file.read(tmp_mem, 4608);
    hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2951
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2951_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2951(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2951_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2951_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2814
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2814_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2814(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2814_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2814_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_103
// Description:	Constant
// Input:
// Output:
//	- name: Constant_103_0	type: float	shape: Shape{5, 5, 32, 1}
void Constant_float_cuda_Constant_103(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_103_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_103_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[3200];
    bin_file.read(tmp_mem, 3200);
    hipMemcpyAsync(output0, tmp_mem, 3200, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2416
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2416_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2416(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2416_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2416_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2476
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2476_0	type: float	shape: Shape{64, 384, 1, 1}
void Constant_float_cuda_Constant_2476(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2476_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2476_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[98304];
    bin_file.read(tmp_mem, 98304);
    hipMemcpyAsync(output0, tmp_mem, 98304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_16
// Description:	Constant
// Input:
// Output:
//	- name: Constant_16_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_16(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_16_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_16_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[2304];
    bin_file.read(tmp_mem, 2304);
    hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_282
// Description:	Constant
// Input:
// Output:
//	- name: Constant_282_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_282(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_282_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_282_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[2304];
    bin_file.read(tmp_mem, 2304);
    hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2092
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2092_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2092(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2092_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2092_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4096];
    bin_file.read(tmp_mem, 4096);
    hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2134
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2134_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2134(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2134_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2134_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4096];
    bin_file.read(tmp_mem, 4096);
    hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2899
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2899_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2899(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2899_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2899_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2260
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2260_0	type: float	shape: Shape{64, 192, 1, 1}
void Constant_float_cuda_Constant_2260(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2260_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2260_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[49152];
    bin_file.read(tmp_mem, 49152);
    hipMemcpyAsync(output0, tmp_mem, 49152, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_804_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2218_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: DepthwiseConv2dNative_805_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2221_0	type: float	shape: Shape{32, 32, 1, 1}
// Output:
//	- name: Convolution_813_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_815_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_813<<<dim3(2, 16, 2), dim3(16, 2, 8), 0, 0>>>(DepthwiseConv2dNative_804_0, Constant_2218_0, Convolution_813_0);
// Convolution_float_float_float_cuda_Convolution_815<<<dim3(2, 16, 2), dim3(16, 2, 8), 0, 0>>>(DepthwiseConv2dNative_805_0, Constant_2221_0, Convolution_815_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Convolution_float_float_float_cuda_Convolution_815 : Convolution_float_float_float_cuda_Convolution_813

// Node name:	Convolution_813
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_804_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2218_0	type: float	shape: Shape{32, 32, 1, 1}
// Output:
//	- name: Convolution_813_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_813_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(16, 2, 8);
    const dim3 gridDim(2, 16, 2);
    const dim3 threadIdx(thread_id % 16, thread_id / 16 % 2, thread_id / 32);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 16, block_id / 32);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 2048);
    {
        float* compute = output0;{
           float compute_local[2];
          
          
          for (int ff_c_init = 0; ff_c_init < 2; ++ff_c_init) {
            compute_local[ff_c_init] = 0.000000e+00f;
          }
          for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
            __syncthreads();
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
              pad_temp_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] = input0[(((((((rc_outer * 16384) + (((int)threadIdx.z) * 2048)) + (((int)threadIdx.y) * 1024)) + (((int)blockIdx.y) * 64)) + ((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 4) * 32)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 15))];
            }
            input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 32)) + (rc_outer * 16)) + ((int)threadIdx.x))];
            __syncthreads();
            for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
              for (int ff_c = 0; ff_c < 2; ++ff_c) {
                compute_local[ff_c] = (compute_local[ff_c] + (pad_temp_shared[(((rc_inner * 32) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x))] * input1_shared[(((((int)threadIdx.z) * 32) + (ff_c * 16)) + rc_inner)]));
              }
            }
          }
          for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 2; ++ff_inner_inner_inner) {
            compute[(((((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 2048)) + (ff_inner_inner_inner * 1024)) + (((int)blockIdx.y) * 64)) + (((int)threadIdx.y) * 32)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x))] = compute_local[ff_inner_inner_inner];
          }
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_43(float* input0, float* input1, float* input2, float* input3, float* output0, float* output1)
{
    __shared__ char shared_buffer[3072];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Convolution_float_float_float_cuda_Convolution_813_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Convolution_float_float_float_cuda_Convolution_813_block_kernel(input2, input3, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_43_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_43<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0, output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_1161_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2407_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3088_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1163_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2413_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3092_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: AvgPool_1157_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1162_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2410_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3090_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Slice_1159_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_1182_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_1184_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Add_1160_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_1183_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_1165_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3087<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1161_0, Constant_2407_0, Constant_3088_0, Relu_1182_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3091<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1163_0, Constant_2413_0, Constant_3092_0, Relu_1184_0);
// Add_float_float_float_cuda_Add_1160<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(AvgPool_1157_0, AvgPool_1157_0, Add_1160_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3089<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1162_0, Constant_2410_0, Constant_3090_0, Relu_1183_0);
// Relu_float_float_cuda_Relu_1165<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(Slice_1159_0, Relu_1165_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3091 : Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3087
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3089 : Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3087

// Node name:	Matched_Pattern_3087
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_1161_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2407_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3088_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_1182_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3087_block_kernel(float* input0, float* input1, float* input2, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(8, 2, 16);
    const dim3 gridDim(2, 8, 4);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 8, block_id / 16);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 1024);
    {
        float* compute = output0;{
           float compute1[1];
          
          
          compute1[0] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[(((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 4096)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 16)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 8192)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 32)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 12288)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 48)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          compute[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))] = max((compute1[0] + input2[((((int)blockIdx.z) * 16) + ((int)threadIdx.z))]), 0.000000e+00f);
        }


    }

}
// Node name:	Add_1160
// Description:	Add
// Input:
//	- name: AvgPool_1157_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: AvgPool_1157_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Add_1160_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Add_float_float_float_cuda_Add_1160_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(32, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = add(input0[blockIdx.x * 512 + threadIdx.x], input1[blockIdx.x * 512 + threadIdx.x]);

}
// Node name:	Relu_1165
// Description:	Relu
// Input:
//	- name: Slice_1159_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_1165_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Relu_float_float_cuda_Relu_1165_block_kernel(float* input0, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(32, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = relu(input0[blockIdx.x * 512 + threadIdx.x]);

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Add_Matched_Pattern_Relu_95(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* output0, float* output1, float* output2, float* output3, float* output4)
{
    __shared__ char shared_buffer[2048];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3087_block_kernel(input0, input1, input2, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3087_block_kernel(input3, input4, input5, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 159)
    {
        Add_float_float_float_cuda_Add_1160_block_kernel(input6, input6, output2, threadIdx.x, blockIdx.x - 128 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 160 && (int)blockIdx.x <= 223)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3087_block_kernel(input7, input8, input9, output3, threadIdx.x, blockIdx.x - 160 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 224 && (int)blockIdx.x <= 255)
    {
        Relu_float_float_cuda_Relu_1165_block_kernel(input10, output4, threadIdx.x, blockIdx.x - 224 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Add_Matched_Pattern_Relu_95_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* output0, float* output1, float* output2, float* output3, float* output4) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Add_Matched_Pattern_Relu_95<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, input6, input7, input8, input9, input10, output0, output1, output2, output3, output4);
}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_912_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_162_0	type: float	shape: Shape{5, 5, 64, 1}
//	- name: BatchNormInference_908_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: DepthwiseConv2dNative_917_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_910_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_917<<<dim3(128, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_912_0, Constant_162_0, DepthwiseConv2dNative_917_0);
// Relu_float_float_cuda_Relu_910<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(BatchNormInference_908_0, Relu_910_0);
// Deduped function map: <src_function_name : deduped_function_name>

// Node name:	DepthwiseConv2dNative_917
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_912_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_162_0	type: float	shape: Shape{5, 5, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_917_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_917_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(128, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 16;
        const int in_width = 16;
        const int in_depth = 64;
        const int filter_height = 5;
        const int filter_width = 5;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 2;
        const int pad_width = 2;
        const int out_height = 16;
        const int out_width = 16;
        const int out_depth = 64;
        const int num_outputs = 16384;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
// Node name:	Relu_910
// Description:	Relu
// Input:
//	- name: BatchNormInference_908_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_910_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Relu_float_float_cuda_Relu_910_block_kernel(float* input0, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(32, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = relu(input0[blockIdx.x * 512 + threadIdx.x]);

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_cuda_DepthwiseConv2dNative_Relu_59(float* input0, float* input1, float* input2, float* output0, float* output1)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 127)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_917_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 159)
    {
        Relu_float_float_cuda_Relu_910_block_kernel(input2, output1, threadIdx.x, blockIdx.x - 128 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_cuda_DepthwiseConv2dNative_Relu_59_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_cuda_DepthwiseConv2dNative_Relu_59<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0, output1);
}
