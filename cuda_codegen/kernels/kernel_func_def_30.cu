#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1)
{
    return x0 + x1;
}
__device__ __forceinline__ float relu(float x0)
{
    return fmaxf(0,x0);
}
// Node name:	Constant_438
// Description:	Constant
// Input:
// Output:
//	- name: Constant_438_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_438(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_438_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_438_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4608];
    bin_file.read(tmp_mem, 4608);
    hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_3128
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3128_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_3128(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_3128_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_3128_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2965
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2965_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2965(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2965_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2965_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_164
// Description:	Constant
// Input:
// Output:
//	- name: Constant_164_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_164(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_164_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_164_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[2304];
    bin_file.read(tmp_mem, 2304);
    hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_3082
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3082_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_3082(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_3082_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_3082_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2083
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2083_0	type: float	shape: Shape{32, 192, 1, 1}
void Constant_float_cuda_Constant_2083(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2083_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2083_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[24576];
    bin_file.read(tmp_mem, 24576);
    hipMemcpyAsync(output0, tmp_mem, 24576, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2827
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2827_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2827(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2827_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2827_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[131072];
    bin_file.read(tmp_mem, 131072);
    hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_463
// Description:	Constant
// Input:
// Output:
//	- name: Constant_463_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_463(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_463_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_463_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1152];
    bin_file.read(tmp_mem, 1152);
    hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2233
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2233_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2233(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2233_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2233_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_1
// Description:	Constant
// Input:
// Output:
//	- name: Constant_1_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_1(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_1_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_1_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1152];
    bin_file.read(tmp_mem, 1152);
    hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2305
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2305_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2305(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2305_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2305_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_1024_0	type: float	shape: Shape{1, 384, 16, 16}
//	- name: Constant_2329_0	type: float	shape: Shape{64, 384, 1, 1}
//	- name: Constant_2332_0	type: float	shape: Shape{64, 384, 1, 1}
// Output:
//	- name: Convolution_1026_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1028_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_1026<<<dim3(1, 16, 4), dim3(16, 1, 16), 0, 0>>>(Relu_1024_0, Constant_2329_0, Convolution_1026_0);
// Convolution_float_float_float_cuda_Convolution_1028<<<dim3(1, 16, 4), dim3(16, 1, 16), 0, 0>>>(Relu_1024_0, Constant_2332_0, Convolution_1028_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Convolution_float_float_float_cuda_Convolution_1028 : Convolution_float_float_float_cuda_Convolution_1026

// Node name:	Convolution_1026
// Description:	Convolution
// Input:
//	- name: Relu_1024_0	type: float	shape: Shape{1, 384, 16, 16}
//	- name: Constant_2329_0	type: float	shape: Shape{64, 384, 1, 1}
// Output:
//	- name: Convolution_1026_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_1026_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(16, 1, 16);
    const dim3 gridDim(1, 16, 4);
    const dim3 threadIdx(thread_id % 16, 0, thread_id / 16);
    const dim3 blockIdx(block_id % 1, block_id / 1 % 16, block_id / 16);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 3072);
    {
        float* compute = output0;{
           float compute_local[1];
          
          
          compute_local[0] = 0.000000e+00f;
          pad_temp_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input0[((((((int)threadIdx.z) * 768) + (((((int)threadIdx.x) * 3) / 16) * 256)) + (((int)blockIdx.y) * 16)) + ((((int)threadIdx.x) * 3) & 15))];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input0[((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 1) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 1) & 15))];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input0[((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 2) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 2) & 15))];
          input1_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input1[(((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3))];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 1)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 2)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 48)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 48) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 48) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 48) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 48) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 48) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 48) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 48) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 48) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 48) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 48) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 48) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 48) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 48) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 48) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 48) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 256)] * input1_shared[((((int)threadIdx.z) * 48) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 272)] * input1_shared[((((int)threadIdx.z) * 48) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 288)] * input1_shared[((((int)threadIdx.z) * 48) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 304)] * input1_shared[((((int)threadIdx.z) * 48) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 320)] * input1_shared[((((int)threadIdx.z) * 48) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 336)] * input1_shared[((((int)threadIdx.z) * 48) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 352)] * input1_shared[((((int)threadIdx.z) * 48) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 368)] * input1_shared[((((int)threadIdx.z) * 48) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 384)] * input1_shared[((((int)threadIdx.z) * 48) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 400)] * input1_shared[((((int)threadIdx.z) * 48) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 416)] * input1_shared[((((int)threadIdx.z) * 48) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 432)] * input1_shared[((((int)threadIdx.z) * 48) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 448)] * input1_shared[((((int)threadIdx.z) * 48) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 464)] * input1_shared[((((int)threadIdx.z) * 48) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 480)] * input1_shared[((((int)threadIdx.z) * 48) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 496)] * input1_shared[((((int)threadIdx.z) * 48) + 31)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 512)] * input1_shared[((((int)threadIdx.z) * 48) + 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 528)] * input1_shared[((((int)threadIdx.z) * 48) + 33)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 544)] * input1_shared[((((int)threadIdx.z) * 48) + 34)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 560)] * input1_shared[((((int)threadIdx.z) * 48) + 35)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 576)] * input1_shared[((((int)threadIdx.z) * 48) + 36)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 592)] * input1_shared[((((int)threadIdx.z) * 48) + 37)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 608)] * input1_shared[((((int)threadIdx.z) * 48) + 38)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 624)] * input1_shared[((((int)threadIdx.z) * 48) + 39)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 640)] * input1_shared[((((int)threadIdx.z) * 48) + 40)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 656)] * input1_shared[((((int)threadIdx.z) * 48) + 41)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 672)] * input1_shared[((((int)threadIdx.z) * 48) + 42)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 688)] * input1_shared[((((int)threadIdx.z) * 48) + 43)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 704)] * input1_shared[((((int)threadIdx.z) * 48) + 44)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 720)] * input1_shared[((((int)threadIdx.z) * 48) + 45)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 736)] * input1_shared[((((int)threadIdx.z) * 48) + 46)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 752)] * input1_shared[((((int)threadIdx.z) * 48) + 47)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input0[(((((((int)threadIdx.z) * 768) + (((((int)threadIdx.x) * 3) / 16) * 256)) + (((int)blockIdx.y) * 16)) + ((((int)threadIdx.x) * 3) & 15)) + 12288)];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input0[(((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 1) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 1) & 15)) + 12288)];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input0[(((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 2) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 2) & 15)) + 12288)];
          input1_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 48)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 49)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 50)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 48)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 48) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 48) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 48) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 48) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 48) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 48) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 48) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 48) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 48) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 48) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 48) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 48) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 48) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 48) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 48) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 256)] * input1_shared[((((int)threadIdx.z) * 48) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 272)] * input1_shared[((((int)threadIdx.z) * 48) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 288)] * input1_shared[((((int)threadIdx.z) * 48) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 304)] * input1_shared[((((int)threadIdx.z) * 48) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 320)] * input1_shared[((((int)threadIdx.z) * 48) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 336)] * input1_shared[((((int)threadIdx.z) * 48) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 352)] * input1_shared[((((int)threadIdx.z) * 48) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 368)] * input1_shared[((((int)threadIdx.z) * 48) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 384)] * input1_shared[((((int)threadIdx.z) * 48) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 400)] * input1_shared[((((int)threadIdx.z) * 48) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 416)] * input1_shared[((((int)threadIdx.z) * 48) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 432)] * input1_shared[((((int)threadIdx.z) * 48) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 448)] * input1_shared[((((int)threadIdx.z) * 48) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 464)] * input1_shared[((((int)threadIdx.z) * 48) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 480)] * input1_shared[((((int)threadIdx.z) * 48) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 496)] * input1_shared[((((int)threadIdx.z) * 48) + 31)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 512)] * input1_shared[((((int)threadIdx.z) * 48) + 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 528)] * input1_shared[((((int)threadIdx.z) * 48) + 33)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 544)] * input1_shared[((((int)threadIdx.z) * 48) + 34)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 560)] * input1_shared[((((int)threadIdx.z) * 48) + 35)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 576)] * input1_shared[((((int)threadIdx.z) * 48) + 36)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 592)] * input1_shared[((((int)threadIdx.z) * 48) + 37)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 608)] * input1_shared[((((int)threadIdx.z) * 48) + 38)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 624)] * input1_shared[((((int)threadIdx.z) * 48) + 39)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 640)] * input1_shared[((((int)threadIdx.z) * 48) + 40)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 656)] * input1_shared[((((int)threadIdx.z) * 48) + 41)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 672)] * input1_shared[((((int)threadIdx.z) * 48) + 42)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 688)] * input1_shared[((((int)threadIdx.z) * 48) + 43)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 704)] * input1_shared[((((int)threadIdx.z) * 48) + 44)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 720)] * input1_shared[((((int)threadIdx.z) * 48) + 45)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 736)] * input1_shared[((((int)threadIdx.z) * 48) + 46)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 752)] * input1_shared[((((int)threadIdx.z) * 48) + 47)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input0[(((((((int)threadIdx.z) * 768) + (((((int)threadIdx.x) * 3) / 16) * 256)) + (((int)blockIdx.y) * 16)) + ((((int)threadIdx.x) * 3) & 15)) + 24576)];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input0[(((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 1) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 1) & 15)) + 24576)];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input0[(((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 2) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 2) & 15)) + 24576)];
          input1_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 96)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 97)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 98)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 48)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 48) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 48) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 48) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 48) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 48) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 48) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 48) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 48) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 48) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 48) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 48) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 48) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 48) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 48) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 48) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 256)] * input1_shared[((((int)threadIdx.z) * 48) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 272)] * input1_shared[((((int)threadIdx.z) * 48) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 288)] * input1_shared[((((int)threadIdx.z) * 48) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 304)] * input1_shared[((((int)threadIdx.z) * 48) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 320)] * input1_shared[((((int)threadIdx.z) * 48) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 336)] * input1_shared[((((int)threadIdx.z) * 48) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 352)] * input1_shared[((((int)threadIdx.z) * 48) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 368)] * input1_shared[((((int)threadIdx.z) * 48) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 384)] * input1_shared[((((int)threadIdx.z) * 48) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 400)] * input1_shared[((((int)threadIdx.z) * 48) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 416)] * input1_shared[((((int)threadIdx.z) * 48) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 432)] * input1_shared[((((int)threadIdx.z) * 48) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 448)] * input1_shared[((((int)threadIdx.z) * 48) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 464)] * input1_shared[((((int)threadIdx.z) * 48) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 480)] * input1_shared[((((int)threadIdx.z) * 48) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 496)] * input1_shared[((((int)threadIdx.z) * 48) + 31)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 512)] * input1_shared[((((int)threadIdx.z) * 48) + 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 528)] * input1_shared[((((int)threadIdx.z) * 48) + 33)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 544)] * input1_shared[((((int)threadIdx.z) * 48) + 34)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 560)] * input1_shared[((((int)threadIdx.z) * 48) + 35)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 576)] * input1_shared[((((int)threadIdx.z) * 48) + 36)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 592)] * input1_shared[((((int)threadIdx.z) * 48) + 37)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 608)] * input1_shared[((((int)threadIdx.z) * 48) + 38)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 624)] * input1_shared[((((int)threadIdx.z) * 48) + 39)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 640)] * input1_shared[((((int)threadIdx.z) * 48) + 40)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 656)] * input1_shared[((((int)threadIdx.z) * 48) + 41)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 672)] * input1_shared[((((int)threadIdx.z) * 48) + 42)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 688)] * input1_shared[((((int)threadIdx.z) * 48) + 43)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 704)] * input1_shared[((((int)threadIdx.z) * 48) + 44)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 720)] * input1_shared[((((int)threadIdx.z) * 48) + 45)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 736)] * input1_shared[((((int)threadIdx.z) * 48) + 46)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 752)] * input1_shared[((((int)threadIdx.z) * 48) + 47)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input0[(((((((int)threadIdx.z) * 768) + (((((int)threadIdx.x) * 3) / 16) * 256)) + (((int)blockIdx.y) * 16)) + ((((int)threadIdx.x) * 3) & 15)) + 36864)];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input0[(((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 1) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 1) & 15)) + 36864)];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input0[(((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 2) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 2) & 15)) + 36864)];
          input1_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 144)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 145)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 146)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 48)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 48) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 48) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 48) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 48) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 48) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 48) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 48) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 48) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 48) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 48) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 48) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 48) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 48) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 48) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 48) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 256)] * input1_shared[((((int)threadIdx.z) * 48) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 272)] * input1_shared[((((int)threadIdx.z) * 48) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 288)] * input1_shared[((((int)threadIdx.z) * 48) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 304)] * input1_shared[((((int)threadIdx.z) * 48) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 320)] * input1_shared[((((int)threadIdx.z) * 48) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 336)] * input1_shared[((((int)threadIdx.z) * 48) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 352)] * input1_shared[((((int)threadIdx.z) * 48) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 368)] * input1_shared[((((int)threadIdx.z) * 48) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 384)] * input1_shared[((((int)threadIdx.z) * 48) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 400)] * input1_shared[((((int)threadIdx.z) * 48) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 416)] * input1_shared[((((int)threadIdx.z) * 48) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 432)] * input1_shared[((((int)threadIdx.z) * 48) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 448)] * input1_shared[((((int)threadIdx.z) * 48) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 464)] * input1_shared[((((int)threadIdx.z) * 48) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 480)] * input1_shared[((((int)threadIdx.z) * 48) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 496)] * input1_shared[((((int)threadIdx.z) * 48) + 31)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 512)] * input1_shared[((((int)threadIdx.z) * 48) + 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 528)] * input1_shared[((((int)threadIdx.z) * 48) + 33)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 544)] * input1_shared[((((int)threadIdx.z) * 48) + 34)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 560)] * input1_shared[((((int)threadIdx.z) * 48) + 35)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 576)] * input1_shared[((((int)threadIdx.z) * 48) + 36)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 592)] * input1_shared[((((int)threadIdx.z) * 48) + 37)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 608)] * input1_shared[((((int)threadIdx.z) * 48) + 38)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 624)] * input1_shared[((((int)threadIdx.z) * 48) + 39)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 640)] * input1_shared[((((int)threadIdx.z) * 48) + 40)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 656)] * input1_shared[((((int)threadIdx.z) * 48) + 41)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 672)] * input1_shared[((((int)threadIdx.z) * 48) + 42)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 688)] * input1_shared[((((int)threadIdx.z) * 48) + 43)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 704)] * input1_shared[((((int)threadIdx.z) * 48) + 44)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 720)] * input1_shared[((((int)threadIdx.z) * 48) + 45)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 736)] * input1_shared[((((int)threadIdx.z) * 48) + 46)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 752)] * input1_shared[((((int)threadIdx.z) * 48) + 47)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input0[(((((((int)threadIdx.z) * 768) + (((((int)threadIdx.x) * 3) / 16) * 256)) + (((int)blockIdx.y) * 16)) + ((((int)threadIdx.x) * 3) & 15)) + 49152)];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input0[(((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 1) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 1) & 15)) + 49152)];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input0[(((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 2) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 2) & 15)) + 49152)];
          input1_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 192)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 193)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 194)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 48)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 48) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 48) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 48) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 48) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 48) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 48) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 48) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 48) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 48) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 48) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 48) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 48) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 48) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 48) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 48) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 256)] * input1_shared[((((int)threadIdx.z) * 48) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 272)] * input1_shared[((((int)threadIdx.z) * 48) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 288)] * input1_shared[((((int)threadIdx.z) * 48) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 304)] * input1_shared[((((int)threadIdx.z) * 48) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 320)] * input1_shared[((((int)threadIdx.z) * 48) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 336)] * input1_shared[((((int)threadIdx.z) * 48) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 352)] * input1_shared[((((int)threadIdx.z) * 48) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 368)] * input1_shared[((((int)threadIdx.z) * 48) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 384)] * input1_shared[((((int)threadIdx.z) * 48) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 400)] * input1_shared[((((int)threadIdx.z) * 48) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 416)] * input1_shared[((((int)threadIdx.z) * 48) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 432)] * input1_shared[((((int)threadIdx.z) * 48) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 448)] * input1_shared[((((int)threadIdx.z) * 48) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 464)] * input1_shared[((((int)threadIdx.z) * 48) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 480)] * input1_shared[((((int)threadIdx.z) * 48) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 496)] * input1_shared[((((int)threadIdx.z) * 48) + 31)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 512)] * input1_shared[((((int)threadIdx.z) * 48) + 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 528)] * input1_shared[((((int)threadIdx.z) * 48) + 33)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 544)] * input1_shared[((((int)threadIdx.z) * 48) + 34)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 560)] * input1_shared[((((int)threadIdx.z) * 48) + 35)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 576)] * input1_shared[((((int)threadIdx.z) * 48) + 36)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 592)] * input1_shared[((((int)threadIdx.z) * 48) + 37)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 608)] * input1_shared[((((int)threadIdx.z) * 48) + 38)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 624)] * input1_shared[((((int)threadIdx.z) * 48) + 39)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 640)] * input1_shared[((((int)threadIdx.z) * 48) + 40)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 656)] * input1_shared[((((int)threadIdx.z) * 48) + 41)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 672)] * input1_shared[((((int)threadIdx.z) * 48) + 42)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 688)] * input1_shared[((((int)threadIdx.z) * 48) + 43)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 704)] * input1_shared[((((int)threadIdx.z) * 48) + 44)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 720)] * input1_shared[((((int)threadIdx.z) * 48) + 45)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 736)] * input1_shared[((((int)threadIdx.z) * 48) + 46)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 752)] * input1_shared[((((int)threadIdx.z) * 48) + 47)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input0[(((((((int)threadIdx.z) * 768) + (((((int)threadIdx.x) * 3) / 16) * 256)) + (((int)blockIdx.y) * 16)) + ((((int)threadIdx.x) * 3) & 15)) + 61440)];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input0[(((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 1) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 1) & 15)) + 61440)];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input0[(((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 2) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 2) & 15)) + 61440)];
          input1_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 240)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 241)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 242)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 48)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 48) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 48) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 48) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 48) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 48) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 48) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 48) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 48) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 48) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 48) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 48) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 48) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 48) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 48) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 48) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 256)] * input1_shared[((((int)threadIdx.z) * 48) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 272)] * input1_shared[((((int)threadIdx.z) * 48) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 288)] * input1_shared[((((int)threadIdx.z) * 48) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 304)] * input1_shared[((((int)threadIdx.z) * 48) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 320)] * input1_shared[((((int)threadIdx.z) * 48) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 336)] * input1_shared[((((int)threadIdx.z) * 48) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 352)] * input1_shared[((((int)threadIdx.z) * 48) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 368)] * input1_shared[((((int)threadIdx.z) * 48) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 384)] * input1_shared[((((int)threadIdx.z) * 48) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 400)] * input1_shared[((((int)threadIdx.z) * 48) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 416)] * input1_shared[((((int)threadIdx.z) * 48) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 432)] * input1_shared[((((int)threadIdx.z) * 48) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 448)] * input1_shared[((((int)threadIdx.z) * 48) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 464)] * input1_shared[((((int)threadIdx.z) * 48) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 480)] * input1_shared[((((int)threadIdx.z) * 48) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 496)] * input1_shared[((((int)threadIdx.z) * 48) + 31)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 512)] * input1_shared[((((int)threadIdx.z) * 48) + 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 528)] * input1_shared[((((int)threadIdx.z) * 48) + 33)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 544)] * input1_shared[((((int)threadIdx.z) * 48) + 34)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 560)] * input1_shared[((((int)threadIdx.z) * 48) + 35)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 576)] * input1_shared[((((int)threadIdx.z) * 48) + 36)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 592)] * input1_shared[((((int)threadIdx.z) * 48) + 37)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 608)] * input1_shared[((((int)threadIdx.z) * 48) + 38)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 624)] * input1_shared[((((int)threadIdx.z) * 48) + 39)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 640)] * input1_shared[((((int)threadIdx.z) * 48) + 40)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 656)] * input1_shared[((((int)threadIdx.z) * 48) + 41)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 672)] * input1_shared[((((int)threadIdx.z) * 48) + 42)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 688)] * input1_shared[((((int)threadIdx.z) * 48) + 43)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 704)] * input1_shared[((((int)threadIdx.z) * 48) + 44)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 720)] * input1_shared[((((int)threadIdx.z) * 48) + 45)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 736)] * input1_shared[((((int)threadIdx.z) * 48) + 46)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 752)] * input1_shared[((((int)threadIdx.z) * 48) + 47)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input0[(((((((int)threadIdx.z) * 768) + (((((int)threadIdx.x) * 3) / 16) * 256)) + (((int)blockIdx.y) * 16)) + ((((int)threadIdx.x) * 3) & 15)) + 73728)];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input0[(((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 1) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 1) & 15)) + 73728)];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input0[(((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 2) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 2) & 15)) + 73728)];
          input1_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 288)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 289)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 290)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 48)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 48) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 48) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 48) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 48) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 48) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 48) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 48) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 48) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 48) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 48) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 48) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 48) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 48) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 48) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 48) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 256)] * input1_shared[((((int)threadIdx.z) * 48) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 272)] * input1_shared[((((int)threadIdx.z) * 48) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 288)] * input1_shared[((((int)threadIdx.z) * 48) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 304)] * input1_shared[((((int)threadIdx.z) * 48) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 320)] * input1_shared[((((int)threadIdx.z) * 48) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 336)] * input1_shared[((((int)threadIdx.z) * 48) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 352)] * input1_shared[((((int)threadIdx.z) * 48) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 368)] * input1_shared[((((int)threadIdx.z) * 48) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 384)] * input1_shared[((((int)threadIdx.z) * 48) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 400)] * input1_shared[((((int)threadIdx.z) * 48) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 416)] * input1_shared[((((int)threadIdx.z) * 48) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 432)] * input1_shared[((((int)threadIdx.z) * 48) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 448)] * input1_shared[((((int)threadIdx.z) * 48) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 464)] * input1_shared[((((int)threadIdx.z) * 48) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 480)] * input1_shared[((((int)threadIdx.z) * 48) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 496)] * input1_shared[((((int)threadIdx.z) * 48) + 31)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 512)] * input1_shared[((((int)threadIdx.z) * 48) + 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 528)] * input1_shared[((((int)threadIdx.z) * 48) + 33)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 544)] * input1_shared[((((int)threadIdx.z) * 48) + 34)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 560)] * input1_shared[((((int)threadIdx.z) * 48) + 35)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 576)] * input1_shared[((((int)threadIdx.z) * 48) + 36)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 592)] * input1_shared[((((int)threadIdx.z) * 48) + 37)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 608)] * input1_shared[((((int)threadIdx.z) * 48) + 38)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 624)] * input1_shared[((((int)threadIdx.z) * 48) + 39)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 640)] * input1_shared[((((int)threadIdx.z) * 48) + 40)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 656)] * input1_shared[((((int)threadIdx.z) * 48) + 41)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 672)] * input1_shared[((((int)threadIdx.z) * 48) + 42)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 688)] * input1_shared[((((int)threadIdx.z) * 48) + 43)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 704)] * input1_shared[((((int)threadIdx.z) * 48) + 44)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 720)] * input1_shared[((((int)threadIdx.z) * 48) + 45)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 736)] * input1_shared[((((int)threadIdx.z) * 48) + 46)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 752)] * input1_shared[((((int)threadIdx.z) * 48) + 47)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input0[(((((((int)threadIdx.z) * 768) + (((((int)threadIdx.x) * 3) / 16) * 256)) + (((int)blockIdx.y) * 16)) + ((((int)threadIdx.x) * 3) & 15)) + 86016)];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input0[(((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 1) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 1) & 15)) + 86016)];
          pad_temp_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input0[(((((((int)threadIdx.z) * 768) + ((((((int)threadIdx.x) * 3) + 2) >> 4) * 256)) + (((int)blockIdx.y) * 16)) + (((((int)threadIdx.x) * 3) + 2) & 15)) + 86016)];
          input1_shared[((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3))] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 336)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 1)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 337)];
          input1_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) + 2)] = input1[((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) + (((int)threadIdx.x) * 3)) + 338)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 48)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 48) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 48) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 48) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 48) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 48) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 48) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 48) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 48) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 48) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 48) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 48) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 48) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 48) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 48) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 48) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 256)] * input1_shared[((((int)threadIdx.z) * 48) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 272)] * input1_shared[((((int)threadIdx.z) * 48) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 288)] * input1_shared[((((int)threadIdx.z) * 48) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 304)] * input1_shared[((((int)threadIdx.z) * 48) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 320)] * input1_shared[((((int)threadIdx.z) * 48) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 336)] * input1_shared[((((int)threadIdx.z) * 48) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 352)] * input1_shared[((((int)threadIdx.z) * 48) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 368)] * input1_shared[((((int)threadIdx.z) * 48) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 384)] * input1_shared[((((int)threadIdx.z) * 48) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 400)] * input1_shared[((((int)threadIdx.z) * 48) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 416)] * input1_shared[((((int)threadIdx.z) * 48) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 432)] * input1_shared[((((int)threadIdx.z) * 48) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 448)] * input1_shared[((((int)threadIdx.z) * 48) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 464)] * input1_shared[((((int)threadIdx.z) * 48) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 480)] * input1_shared[((((int)threadIdx.z) * 48) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 496)] * input1_shared[((((int)threadIdx.z) * 48) + 31)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 512)] * input1_shared[((((int)threadIdx.z) * 48) + 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 528)] * input1_shared[((((int)threadIdx.z) * 48) + 33)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 544)] * input1_shared[((((int)threadIdx.z) * 48) + 34)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 560)] * input1_shared[((((int)threadIdx.z) * 48) + 35)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 576)] * input1_shared[((((int)threadIdx.z) * 48) + 36)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 592)] * input1_shared[((((int)threadIdx.z) * 48) + 37)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 608)] * input1_shared[((((int)threadIdx.z) * 48) + 38)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 624)] * input1_shared[((((int)threadIdx.z) * 48) + 39)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 640)] * input1_shared[((((int)threadIdx.z) * 48) + 40)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 656)] * input1_shared[((((int)threadIdx.z) * 48) + 41)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 672)] * input1_shared[((((int)threadIdx.z) * 48) + 42)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 688)] * input1_shared[((((int)threadIdx.z) * 48) + 43)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 704)] * input1_shared[((((int)threadIdx.z) * 48) + 44)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 720)] * input1_shared[((((int)threadIdx.z) * 48) + 45)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 736)] * input1_shared[((((int)threadIdx.z) * 48) + 46)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 752)] * input1_shared[((((int)threadIdx.z) * 48) + 47)]));
          compute[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x))] = compute_local[0];
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_cuda_Convolution_Convolution_74(float* input0, float* input1, float* input2, float* output0, float* output1)
{
    __shared__ char shared_buffer[6144];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Convolution_float_float_float_cuda_Convolution_1026_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Convolution_float_float_float_cuda_Convolution_1026_block_kernel(input0, input2, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_cuda_Convolution_Convolution_74_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_cuda_Convolution_Convolution_74<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0, output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_977_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2308_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3064_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_978_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2311_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3066_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1000_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2314_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: DepthwiseConv2dNative_1002_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2320_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: DepthwiseConv2dNative_1001_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2317_0	type: float	shape: Shape{64, 64, 1, 1}
// Output:
//	- name: Relu_998_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_999_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1006_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1010_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1008_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3063<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_977_0, Constant_2308_0, Constant_3064_0, Relu_998_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3065<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_978_0, Constant_2311_0, Constant_3066_0, Relu_999_0);
// Convolution_float_float_float_cuda_Convolution_1006<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1000_0, Constant_2314_0, Convolution_1006_0);
// Convolution_float_float_float_cuda_Convolution_1010<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1002_0, Constant_2320_0, Convolution_1010_0);
// Convolution_float_float_float_cuda_Convolution_1008<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1001_0, Constant_2317_0, Convolution_1008_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3065 : Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3063
// Convolution_float_float_float_cuda_Convolution_1010 : Convolution_float_float_float_cuda_Convolution_1006
// Convolution_float_float_float_cuda_Convolution_1008 : Convolution_float_float_float_cuda_Convolution_1006

// Node name:	Matched_Pattern_3063
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_977_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2308_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3064_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_998_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3063_block_kernel(float* input0, float* input1, float* input2, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(8, 2, 16);
    const dim3 gridDim(2, 8, 4);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 8, block_id / 16);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 1024);
    {
        float* compute = output0;{
           float compute1[1];
          
          
          compute1[0] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[(((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 4096)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 16)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 8192)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 32)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 12288)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 48)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          compute[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))] = max((compute1[0] + input2[((((int)blockIdx.z) * 16) + ((int)threadIdx.z))]), 0.000000e+00f);
        }


    }

}
// Node name:	Convolution_1006
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_1000_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2314_0	type: float	shape: Shape{64, 64, 1, 1}
// Output:
//	- name: Convolution_1006_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_1006_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(8, 2, 16);
    const dim3 gridDim(2, 8, 4);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 8, block_id / 16);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 1024);
    {
        float* compute = output0;{
           float compute_local[1];
          
          
          compute_local[0] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[(((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 4096)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 16)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 8192)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 32)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 12288)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 48)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          compute[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))] = compute_local[0];
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Convolution_Convolution_Convolution_70(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* input11, float* output0, float* output1, float* output2, float* output3, float* output4)
{
    __shared__ char shared_buffer[2048];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3063_block_kernel(input0, input1, input2, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3063_block_kernel(input3, input4, input5, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191)
    {
        Convolution_float_float_float_cuda_Convolution_1006_block_kernel(input6, input7, output2, threadIdx.x, blockIdx.x - 128 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 192 && (int)blockIdx.x <= 255)
    {
        Convolution_float_float_float_cuda_Convolution_1006_block_kernel(input8, input9, output3, threadIdx.x, blockIdx.x - 192 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 256 && (int)blockIdx.x <= 319)
    {
        Convolution_float_float_float_cuda_Convolution_1006_block_kernel(input10, input11, output4, threadIdx.x, blockIdx.x - 256 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Convolution_Convolution_Convolution_70_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* input11, float* output0, float* output1, float* output2, float* output3, float* output4) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Convolution_Convolution_Convolution_70<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, input6, input7, input8, input9, input10, input11, output0, output1, output2, output3, output4);
}
// Node name:	 BlockFusion
// Input:
//	- name: Convolution_1414_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2938_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1416_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2937_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: BatchNormInference_1417_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1420_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1418_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// FusedKernel_float_float_float_float_cuda_Add_Relu_53<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1414_0, Constant_2938_0, Relu_1420_0, BatchNormInference_1417_0);
// Add_float_float_float_cuda_Add_2547<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1416_0, Constant_2937_0, BatchNormInference_1418_0);
// Deduped function map: <src_function_name : deduped_function_name>

// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_1414_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2938_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1420_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1417_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Add_float_float_float_cuda_Add_2544<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1414_0, Constant_2938_0, BatchNormInference_1417_0);
// Relu_float_float_cuda_Relu_1420<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(BatchNormInference_1417_0, Relu_1420_0);
__device__ __noinline__ void FusedKernel_float_float_float_float_cuda_Add_Relu_53_block_kernel(float* input0, float* input1, float* output0, float* output1, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(16, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    int tid = blockIdx.x * 512 + threadIdx.x;
    float temp0 = add(input0[tid], input1[tid]);
    float temp1 = relu(temp0);
    output1[tid] = temp0;
    output0[tid] = temp1;

}
// Node name:	Add_2547
// Description:	Add
// Input:
//	- name: Convolution_1416_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2937_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: BatchNormInference_1418_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void Add_float_float_float_cuda_Add_2547_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(16, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = add(input0[blockIdx.x * 512 + threadIdx.x], input1[blockIdx.x * 512 + threadIdx.x]);

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_cuda_fused_kernel_Add_131(float* input0, float* input1, float* input2, float* input3, float* output0, float* output1, float* output2)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15)
    {
        FusedKernel_float_float_float_float_cuda_Add_Relu_53_block_kernel(input0, input1, output1, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31)
    {
        Add_float_float_float_cuda_Add_2547_block_kernel(input2, input3, output2, threadIdx.x, blockIdx.x - 16 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_cuda_fused_kernel_Add_131_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* output0, float* output1, float* output2) {
    BlockFusionKernel_float_float_float_float_float_float_float_cuda_fused_kernel_Add_131<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0, output1, output2);
}
// Node name:	 BlockFusion
// Input:
//	- name: Constant_2806_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1328_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: BatchNormInference_1269_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2766_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1330_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Slice_1284_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Add_1333_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Add_1334_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// FusedKernel_float_float_float_float_cuda_Add_Add_48<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1328_0, Constant_2806_0, BatchNormInference_1269_0, Add_1333_0);
// FusedKernel_float_float_float_float_cuda_Add_Add_49<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1330_0, Constant_2766_0, Slice_1284_0, Add_1334_0);
// Deduped function map: <src_function_name : deduped_function_name>
// FusedKernel_float_float_float_float_cuda_Add_Add_49 : FusedKernel_float_float_float_float_cuda_Add_Add_48

// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_1328_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2806_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: BatchNormInference_1269_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Add_1333_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Add_float_float_float_cuda_Add_2505<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1328_0, Constant_2806_0, BatchNormInference_1331_0);
// Add_float_float_float_cuda_Add_1333<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(BatchNormInference_1331_0, BatchNormInference_1269_0, Add_1333_0);
__device__ __noinline__ void FusedKernel_float_float_float_float_cuda_Add_Add_48_block_kernel(float* input0, float* input1, float* input2, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(32, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    int tid = blockIdx.x * 512 + threadIdx.x;
    float temp0 = add(input0[tid], input1[tid]);
    float temp1 = add(temp0, input2[tid]);
    output0[tid] = temp1;

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_118(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* output0, float* output1)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 31)
    {
        FusedKernel_float_float_float_float_cuda_Add_Add_48_block_kernel(input1, input0, input2, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 63)
    {
        FusedKernel_float_float_float_float_cuda_Add_Add_48_block_kernel(input4, input3, input5, output1, threadIdx.x, blockIdx.x - 32 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_118_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_118<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, output0, output1);
}
