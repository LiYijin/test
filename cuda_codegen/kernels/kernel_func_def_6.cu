#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1)
{
    return x0 + x1;
}
__device__ __forceinline__ float relu(float x0)
{
    return fmaxf(0,x0);
}
// Node name:	 BlockFusion
// Input:
//	- name: AvgPool_646_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: BatchNormInference_578_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Relu_647_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_332_0	type: float	shape: Shape{5, 5, 32, 1}
//	- name: Constant_123_0	type: float	shape: Shape{3, 3, 32, 1}
//	- name: Relu_671_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_452_0	type: float	shape: Shape{3, 3, 32, 1}
//	- name: Relu_672_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_88_0	type: float	shape: Shape{3, 3, 32, 1}
//	- name: Relu_670_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_56_0	type: float	shape: Shape{5, 5, 32, 1}
// Output:
//	- name: Add_652_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_653_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_654_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_676_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_677_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_675_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Add_float_float_float_cuda_Add_652<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(AvgPool_646_0, BatchNormInference_578_0, Add_652_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_653<<<dim3(256, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_647_0, Constant_332_0, DepthwiseConv2dNative_653_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_654<<<dim3(256, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_647_0, Constant_123_0, DepthwiseConv2dNative_654_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_676<<<dim3(256, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_671_0, Constant_452_0, DepthwiseConv2dNative_676_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_677<<<dim3(256, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_672_0, Constant_88_0, DepthwiseConv2dNative_677_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_675<<<dim3(256, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_670_0, Constant_56_0, DepthwiseConv2dNative_675_0);
// Deduped function map: <src_function_name : deduped_function_name>
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_676 : DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_654
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_677 : DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_654
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_675 : DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_653

// Node name:	Add_652
// Description:	Add
// Input:
//	- name: AvgPool_646_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: BatchNormInference_578_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Add_652_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void Add_float_float_float_cuda_Add_652_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = add(input0[blockIdx.x * 512 + threadIdx.x], input1[blockIdx.x * 512 + threadIdx.x]);

}
// Node name:	DepthwiseConv2dNative_653
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_647_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_332_0	type: float	shape: Shape{5, 5, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_653_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_653_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(256, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 32;
        const int in_width = 32;
        const int in_depth = 32;
        const int filter_height = 5;
        const int filter_width = 5;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 2;
        const int pad_width = 2;
        const int out_height = 32;
        const int out_width = 32;
        const int out_depth = 32;
        const int num_outputs = 32768;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
// Node name:	DepthwiseConv2dNative_654
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_647_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_123_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_654_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_654_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(256, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 32;
        const int in_width = 32;
        const int in_depth = 32;
        const int filter_height = 3;
        const int filter_width = 3;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 1;
        const int pad_width = 1;
        const int out_height = 32;
        const int out_width = 32;
        const int out_depth = 32;
        const int num_outputs = 32768;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_22(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* output0, float* output1, float* output2, float* output3, float* output4, float* output5)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Add_float_float_float_cuda_Add_652_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 319)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_653_block_kernel(input2, input3, output1, threadIdx.x, blockIdx.x - 64 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 320 && (int)blockIdx.x <= 575)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_654_block_kernel(input2, input4, output2, threadIdx.x, blockIdx.x - 320 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 576 && (int)blockIdx.x <= 831)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_654_block_kernel(input5, input6, output3, threadIdx.x, blockIdx.x - 576 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 832 && (int)blockIdx.x <= 1087)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_654_block_kernel(input7, input8, output4, threadIdx.x, blockIdx.x - 832 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 1088 && (int)blockIdx.x <= 1343)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_653_block_kernel(input9, input10, output5, threadIdx.x, blockIdx.x - 1088 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_22_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* output0, float* output1, float* output2, float* output3, float* output4, float* output5) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_22<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, input6, input7, input8, input9, input10, output0, output1, output2, output3, output4, output5);
}
// Node name:	Constant_2407
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2407_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2407(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2407_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2407_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2404
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2404_0	type: float	shape: Shape{64, 384, 1, 1}
void Constant_float_cuda_Constant_2404(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2404_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2404_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[98304];
    bin_file.read(tmp_mem, 98304);
    hipMemcpyAsync(output0, tmp_mem, 98304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_169
// Description:	Constant
// Input:
// Output:
//	- name: Constant_169_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_169(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_169_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_169_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1152];
    bin_file.read(tmp_mem, 1152);
    hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_3180
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3180_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_3180(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_3180_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_3180_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2437
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2437_0	type: float	shape: Shape{64, 384, 1, 1}
void Constant_float_cuda_Constant_2437(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2437_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2437_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[98304];
    bin_file.read(tmp_mem, 98304);
    hipMemcpyAsync(output0, tmp_mem, 98304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2855
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2855_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2855(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2855_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2855_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[131072];
    bin_file.read(tmp_mem, 131072);
    hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2296
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2296_0	type: float	shape: Shape{64, 256, 1, 1}
void Constant_float_cuda_Constant_2296(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2296_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2296_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_3048
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3048_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_3048(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_3048_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_3048_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_3148
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3148_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_3148(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_3148_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_3148_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2158
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2158_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2158(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2158_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2158_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4096];
    bin_file.read(tmp_mem, 4096);
    hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_961_0	type: float	shape: Shape{1, 256, 16, 16}
//	- name: Constant_2293_0	type: float	shape: Shape{64, 256, 1, 1}
//	- name: Constant_2296_0	type: float	shape: Shape{64, 256, 1, 1}
// Output:
//	- name: Convolution_963_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_965_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_963<<<dim3(1, 16, 4), dim3(16, 1, 16), 0, 0>>>(Relu_961_0, Constant_2293_0, Convolution_963_0);
// Convolution_float_float_float_cuda_Convolution_965<<<dim3(1, 16, 4), dim3(16, 1, 16), 0, 0>>>(Relu_961_0, Constant_2296_0, Convolution_965_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Convolution_float_float_float_cuda_Convolution_965 : Convolution_float_float_float_cuda_Convolution_963

// Node name:	Convolution_963
// Description:	Convolution
// Input:
//	- name: Relu_961_0	type: float	shape: Shape{1, 256, 16, 16}
//	- name: Constant_2293_0	type: float	shape: Shape{64, 256, 1, 1}
// Output:
//	- name: Convolution_963_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_963_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(16, 1, 16);
    const dim3 gridDim(1, 16, 4);
    const dim3 threadIdx(thread_id % 16, 0, thread_id / 16);
    const dim3 blockIdx(block_id % 1, block_id / 1 % 16, block_id / 16);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 1024);
    {
        float* compute = output0;{
           float compute_local[1];
          
          
          compute_local[0] = 0.000000e+00f;
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[(((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x))];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[(((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x))];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 4096)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 16)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 8192)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 32)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 12288)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 48)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 16384)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 64)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 20480)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 80)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 24576)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 96)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 28672)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 112)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 32768)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 128)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 36864)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 144)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 40960)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 160)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 45056)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 176)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 49152)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 192)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 53248)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 208)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 57344)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 224)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x)) + 61440)];
          input1_shared[((((int)threadIdx.z) * 16) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((int)threadIdx.x)) + 240)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((int)threadIdx.x)] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          compute[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 16)) + ((int)threadIdx.x))] = compute_local[0];
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_cuda_Convolution_Convolution_65(float* input0, float* input1, float* input2, float* output0, float* output1)
{
    __shared__ char shared_buffer[2048];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Convolution_float_float_float_cuda_Convolution_963_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Convolution_float_float_float_cuda_Convolution_963_block_kernel(input0, input2, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_cuda_Convolution_Convolution_65_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_cuda_Convolution_Convolution_65<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0, output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_1475_0	type: float	shape: Shape{1, 768, 8, 8}
//	- name: Constant_2578_0	type: float	shape: Shape{128, 768, 1, 1}
//	- name: Constant_2581_0	type: float	shape: Shape{128, 768, 1, 1}
// Output:
//	- name: Convolution_1477_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1479_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_1477<<<dim3(2, 2, 8), dim3(4, 4, 16), 0, 0>>>(Relu_1475_0, Constant_2578_0, Convolution_1477_0);
// Convolution_float_float_float_cuda_Convolution_1479<<<dim3(2, 2, 8), dim3(4, 4, 16), 0, 0>>>(Relu_1475_0, Constant_2581_0, Convolution_1479_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Convolution_float_float_float_cuda_Convolution_1479 : Convolution_float_float_float_cuda_Convolution_1477

// Node name:	Convolution_1477
// Description:	Convolution
// Input:
//	- name: Relu_1475_0	type: float	shape: Shape{1, 768, 8, 8}
//	- name: Constant_2578_0	type: float	shape: Shape{128, 768, 1, 1}
// Output:
//	- name: Convolution_1477_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_1477_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(4, 4, 16);
    const dim3 gridDim(2, 2, 8);
    const dim3 threadIdx(thread_id % 4, thread_id / 4 % 4, thread_id / 16);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 2, block_id / 4);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 2048);
    {
        float* compute = output0;{
           float compute_local[1];
          
          
          compute_local[0] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2))];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3))];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 2048)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 2048)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 32)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 33)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 4096)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 4096)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 64)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 65)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 6144)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 6144)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 96)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 97)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 8192)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 8192)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 128)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 129)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 10240)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 10240)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 160)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 161)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 12288)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 12288)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 192)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 193)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 14336)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 14336)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 224)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 225)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 16384)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 16384)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 256)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 257)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 18432)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 18432)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 288)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 289)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 20480)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 20480)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 320)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 321)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 22528)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 22528)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 352)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 353)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 24576)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 24576)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 384)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 385)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 26624)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 26624)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 416)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 417)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 28672)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 28672)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 448)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 449)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 30720)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 30720)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 480)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 481)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 32768)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 32768)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 512)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 513)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 34816)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 34816)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 544)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 545)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 36864)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 36864)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 576)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 577)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 38912)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 38912)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 608)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 609)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 40960)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 40960)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 640)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 641)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 43008)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 43008)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 672)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 673)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 45056)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 45056)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 704)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 705)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 1)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 47104)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) >> 2) * 64)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) >> 2)) & 3) * 8)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)) + 47104)];
          input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 736)];
          input1_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 737)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 4) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute[((((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 8)) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x))] = compute_local[0];
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_cuda_Convolution_Convolution_139(float* input0, float* input1, float* input2, float* output0, float* output1)
{
    __shared__ char shared_buffer[4096];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 31)
    {
        Convolution_float_float_float_cuda_Convolution_1477_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 63)
    {
        Convolution_float_float_float_cuda_Convolution_1477_block_kernel(input0, input2, output1, threadIdx.x, blockIdx.x - 32 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_cuda_Convolution_Convolution_139_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_cuda_Convolution_Convolution_139<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0, output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: Slice_895_0	type: float	shape: Shape{1, 64, 32, 32}
//	- name: AvgPool_896_0	type: float	shape: Shape{1, 192, 16, 16}
//	- name: Constant_1853_0	type: float	shape: Shape{32, 192, 1, 1}
// Output:
//	- name: Relu_899_0	type: float	shape: Shape{1, 64, 32, 32}
//	- name: Convolution_901_0	type: float	shape: Shape{1, 32, 16, 16}
// Fused functions:
// Relu_float_float_cuda_Relu_899<<<dim3(128, 1, 1), dim3(512, 1, 1), 0, 0>>>(Slice_895_0, Relu_899_0);
// Convolution_float_float_float_cuda_Convolution_901<<<dim3(2, 8, 4), dim3(8, 2, 8), 0, 0>>>(AvgPool_896_0, Constant_1853_0, Convolution_901_0);
// Deduped function map: <src_function_name : deduped_function_name>

// Node name:	Relu_899
// Description:	Relu
// Input:
//	- name: Slice_895_0	type: float	shape: Shape{1, 64, 32, 32}
// Output:
//	- name: Relu_899_0	type: float	shape: Shape{1, 64, 32, 32}
__device__ __noinline__ void Relu_float_float_cuda_Relu_899_block_kernel(float* input0, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(128, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = relu(input0[blockIdx.x * 512 + threadIdx.x]);

}
// Node name:	Convolution_901
// Description:	Convolution
// Input:
//	- name: AvgPool_896_0	type: float	shape: Shape{1, 192, 16, 16}
//	- name: Constant_1853_0	type: float	shape: Shape{32, 192, 1, 1}
// Output:
//	- name: Convolution_901_0	type: float	shape: Shape{1, 32, 16, 16}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_901_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(8, 2, 8);
    const dim3 gridDim(2, 8, 4);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 8, block_id / 16);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 3072);
    {
        float* compute = output0;{
           float compute_local[1];
          
          
          compute_local[0] = 0.000000e+00f;
          #pragma unroll
          for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
            __syncthreads();
            #pragma unroll
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
              pad_temp_shared[((((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 48)) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] = input0[((((((((rc_outer * 12288) + (((int)threadIdx.z) * 1536)) + (((int)threadIdx.y) * 768)) + ((((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 4) * 256)) + (((int)blockIdx.y) * 32)) + (((((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 15) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 7))];
            }
            #pragma unroll
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
              input1_shared[((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)] = input1[((((((((int)blockIdx.z) * 1536) + (((int)threadIdx.z) * 192)) + (rc_outer * 48)) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)];
            }
            __syncthreads();
            #pragma unroll
            for (int rc_inner = 0; rc_inner < 48; ++rc_inner) {
              compute_local[0] = (compute_local[0] + (pad_temp_shared[(((rc_inner * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 48) + rc_inner)]));
            }
          }
          compute[((((((((int)blockIdx.z) * 2048) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))] = compute_local[0];
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_cuda_Relu_Convolution_56(float* input0, float* input1, float* input2, float* output0, float* output1)
{
    __shared__ char shared_buffer[4608];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 127)
    {
        Relu_float_float_cuda_Relu_899_block_kernel(input0, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191)
    {
        Convolution_float_float_float_cuda_Convolution_901_block_kernel(input1, input2, output1, threadIdx.x, blockIdx.x - 128 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_cuda_Relu_Convolution_56_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_cuda_Relu_Convolution_56<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0, output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: Constant_2788_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_877_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: BatchNormInference_817_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2863_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_879_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Slice_833_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Add_882_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Add_883_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// FusedKernel_float_float_float_float_cuda_Add_Add_23<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_877_0, Constant_2788_0, BatchNormInference_817_0, Add_882_0);
// FusedKernel_float_float_float_float_cuda_Add_Add_24<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_879_0, Constant_2863_0, Slice_833_0, Add_883_0);
// Deduped function map: <src_function_name : deduped_function_name>
// FusedKernel_float_float_float_float_cuda_Add_Add_24 : FusedKernel_float_float_float_float_cuda_Add_Add_23

// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_877_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2788_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: BatchNormInference_817_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Add_882_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Add_float_float_float_cuda_Add_2256<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_877_0, Constant_2788_0, BatchNormInference_880_0);
// Add_float_float_float_cuda_Add_882<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(BatchNormInference_880_0, BatchNormInference_817_0, Add_882_0);
__device__ __noinline__ void FusedKernel_float_float_float_float_cuda_Add_Add_23_block_kernel(float* input0, float* input1, float* input2, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    int tid = blockIdx.x * 512 + threadIdx.x;
    float temp0 = add(input0[tid], input1[tid]);
    float temp1 = add(temp0, input2[tid]);
    output0[tid] = temp1;

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_53(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* output0, float* output1)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        FusedKernel_float_float_float_float_cuda_Add_Add_23_block_kernel(input1, input0, input2, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        FusedKernel_float_float_float_float_cuda_Add_Add_23_block_kernel(input4, input3, input5, output1, threadIdx.x, blockIdx.x - 64 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_53_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_53<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, output0, output1);
}
