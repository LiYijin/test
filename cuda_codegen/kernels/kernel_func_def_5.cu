#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float relu(float x0)
{
    return fmaxf(0,x0);
}
__device__ __forceinline__ float add(float x0, float x1)
{
    return x0 + x1;
}
// Node name:	Constant_364
// Description:	Constant
// Input:
// Output:
//	- name: Constant_364_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_364(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_364_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_364_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1152];
    bin_file.read(tmp_mem, 1152);
    hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2888
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2888_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2888(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2888_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2888_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_146
// Description:	Constant
// Input:
// Output:
//	- name: Constant_146_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_146(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_146_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_146_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4608];
    bin_file.read(tmp_mem, 4608);
    hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2728
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2728_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2728(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2728_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2728_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2933
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2933_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2933(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2933_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2933_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2398
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2398_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2398(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2398_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2398_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2173
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2173_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2173(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2173_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2173_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4096];
    bin_file.read(tmp_mem, 4096);
    hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_131
// Description:	Constant
// Input:
// Output:
//	- name: Constant_131_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_131(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_131_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_131_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[2304];
    bin_file.read(tmp_mem, 2304);
    hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_379
// Description:	Constant
// Input:
// Output:
//	- name: Constant_379_0	type: float	shape: Shape{5, 5, 32, 1}
void Constant_float_cuda_Constant_379(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_379_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_379_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[3200];
    bin_file.read(tmp_mem, 3200);
    hipMemcpyAsync(output0, tmp_mem, 3200, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_3062
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3062_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_3062(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_3062_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_3062_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_1314_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_292_0	type: float	shape: Shape{5, 5, 64, 1}
//	- name: Relu_1315_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_416_0	type: float	shape: Shape{3, 3, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_1322_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1323_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1322<<<dim3(128, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1314_0, Constant_292_0, DepthwiseConv2dNative_1322_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1323<<<dim3(128, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1315_0, Constant_416_0, DepthwiseConv2dNative_1323_0);
// Deduped function map: <src_function_name : deduped_function_name>

// Node name:	DepthwiseConv2dNative_1322
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1314_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_292_0	type: float	shape: Shape{5, 5, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_1322_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1322_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(128, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 16;
        const int in_width = 16;
        const int in_depth = 64;
        const int filter_height = 5;
        const int filter_width = 5;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 2;
        const int pad_width = 2;
        const int out_height = 16;
        const int out_width = 16;
        const int out_depth = 64;
        const int num_outputs = 16384;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
// Node name:	DepthwiseConv2dNative_1323
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1315_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_416_0	type: float	shape: Shape{3, 3, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_1323_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1323_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(128, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 16;
        const int in_width = 16;
        const int in_depth = 64;
        const int filter_height = 3;
        const int filter_width = 3;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 1;
        const int pad_width = 1;
        const int out_height = 16;
        const int out_width = 16;
        const int out_depth = 64;
        const int num_outputs = 16384;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_116(float* input0, float* input1, float* input2, float* input3, float* output0, float* output1)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 127)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1322_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 255)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1323_block_kernel(input2, input3, output1, threadIdx.x, blockIdx.x - 128 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_116_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_116<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0, output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: Slice_1671_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: AvgPool_1673_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1678_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2698_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3174_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1676_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2692_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3170_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1677_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2695_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3172_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1675_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Add_1679_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1700_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1698_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1699_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Relu_float_float_cuda_Relu_1675<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Slice_1671_0, Relu_1675_0);
// Add_float_float_float_cuda_Add_1679<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(AvgPool_1673_0, AvgPool_1673_0, Add_1679_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3173<<<dim3(1, 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1678_0, Constant_2698_0, Constant_3174_0, Relu_1700_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3169<<<dim3(1, 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1676_0, Constant_2692_0, Constant_3170_0, Relu_1698_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3171<<<dim3(1, 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1677_0, Constant_2695_0, Constant_3172_0, Relu_1699_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3169 : Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3173
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3171 : Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3173

// Node name:	Relu_1675
// Description:	Relu
// Input:
//	- name: Slice_1671_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1675_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void Relu_float_float_cuda_Relu_1675_block_kernel(float* input0, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(16, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = relu(input0[blockIdx.x * 512 + threadIdx.x]);

}
// Node name:	Add_1679
// Description:	Add
// Input:
//	- name: AvgPool_1673_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: AvgPool_1673_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1679_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void Add_float_float_float_cuda_Add_1679_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(16, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = add(input0[blockIdx.x * 512 + threadIdx.x], input1[blockIdx.x * 512 + threadIdx.x]);

}
// Node name:	Matched_Pattern_3173
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_1678_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2698_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3174_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1700_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3173_block_kernel(float* input0, float* input1, float* input2, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(8, 2, 8);
    const dim3 gridDim(1, 4, 16);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 1, block_id / 1 % 4, block_id / 4);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 1024);
    {
        float* compute = output0;{
           float compute1[1];
          
          
          compute1[0] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2))];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1024)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1025)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 16)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2048)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2049)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 32)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 3072)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 3073)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 48)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 4096)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 4097)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 64)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 5120)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 5121)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 80)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 6144)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 6145)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 96)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 7168)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 7169)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 112)];
          __syncthreads();
          compute1[0] = (compute1[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute1[0] = (compute1[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          compute[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = max((compute1[0] + input2[((((int)blockIdx.z) * 8) + ((int)threadIdx.z))]), 0.000000e+00f);
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Add_Matched_Pattern_Matched_Pattern_Matched_Pattern_169(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* output0, float* output1, float* output2, float* output3, float* output4)
{
    __shared__ char shared_buffer[1536];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15)
    {
        Relu_float_float_cuda_Relu_1675_block_kernel(input0, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31)
    {
        Add_float_float_float_cuda_Add_1679_block_kernel(input1, input1, output1, threadIdx.x, blockIdx.x - 16 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 95)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3173_block_kernel(input2, input3, input4, output2, threadIdx.x, blockIdx.x - 32 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 96 && (int)blockIdx.x <= 159)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3173_block_kernel(input5, input6, input7, output3, threadIdx.x, blockIdx.x - 96 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 160 && (int)blockIdx.x <= 223)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3173_block_kernel(input8, input9, input10, output4, threadIdx.x, blockIdx.x - 160 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Add_Matched_Pattern_Matched_Pattern_Matched_Pattern_169_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* output0, float* output1, float* output2, float* output3, float* output4) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Add_Matched_Pattern_Matched_Pattern_Matched_Pattern_169<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, input6, input7, input8, input9, input10, output0, output1, output2, output3, output4);
}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_1283_0	type: float	shape: Shape{1, 128, 16, 16}
//	- name: Constant_63_0	type: float	shape: Shape{5, 5, 128, 1}
//	- name: Constant_361_0	type: float	shape: Shape{7, 7, 128, 1}
//	- name: Constant_217_0	type: float	shape: Shape{7, 7, 128, 1}
//	- name: Convolution_1280_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2928_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: DepthwiseConv2dNative_1287_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1285_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1286_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1282_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1287<<<dim3(64, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1283_0, Constant_63_0, DepthwiseConv2dNative_1287_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1285<<<dim3(64, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1283_0, Constant_361_0, DepthwiseConv2dNative_1285_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1286<<<dim3(64, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1283_0, Constant_217_0, DepthwiseConv2dNative_1286_0);
// Add_float_float_float_cuda_Add_2478<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1280_0, Constant_2928_0, BatchNormInference_1282_0);
// Deduped function map: <src_function_name : deduped_function_name>
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1286 : DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1285

// Node name:	DepthwiseConv2dNative_1287
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1283_0	type: float	shape: Shape{1, 128, 16, 16}
//	- name: Constant_63_0	type: float	shape: Shape{5, 5, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1287_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1287_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 16;
        const int in_width = 16;
        const int in_depth = 128;
        const int filter_height = 5;
        const int filter_width = 5;
        const int depth_multiplier = 1;
        const int stride = 2;
        const int pad_height = 1;
        const int pad_width = 1;
        const int out_height = 8;
        const int out_width = 8;
        const int out_depth = 128;
        const int num_outputs = 8192;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
// Node name:	DepthwiseConv2dNative_1285
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1283_0	type: float	shape: Shape{1, 128, 16, 16}
//	- name: Constant_361_0	type: float	shape: Shape{7, 7, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1285_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1285_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 16;
        const int in_width = 16;
        const int in_depth = 128;
        const int filter_height = 7;
        const int filter_width = 7;
        const int depth_multiplier = 1;
        const int stride = 2;
        const int pad_height = 2;
        const int pad_width = 2;
        const int out_height = 8;
        const int out_width = 8;
        const int out_depth = 128;
        const int num_outputs = 8192;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
// Node name:	Add_2478
// Description:	Add
// Input:
//	- name: Convolution_1280_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2928_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: BatchNormInference_1282_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Add_float_float_float_cuda_Add_2478_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(32, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = add(input0[blockIdx.x * 512 + threadIdx.x], input1[blockIdx.x * 512 + threadIdx.x]);

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_111(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* output0, float* output1, float* output2, float* output3)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1287_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1285_block_kernel(input0, input2, output1, threadIdx.x, blockIdx.x - 64 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1285_block_kernel(input0, input3, output2, threadIdx.x, blockIdx.x - 128 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 192 && (int)blockIdx.x <= 223)
    {
        Add_float_float_float_cuda_Add_2478_block_kernel(input4, input5, output3, threadIdx.x, blockIdx.x - 192 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_111_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* output0, float* output1, float* output2, float* output3) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_111<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, output0, output1, output2, output3);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_736_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2173_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: DepthwiseConv2dNative_737_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2176_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: DepthwiseConv2dNative_738_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2179_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: DepthwiseConv2dNative_721_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2167_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: Constant_3022_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_722_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2170_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: Constant_3024_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Convolution_742_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_744_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_746_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Relu_739_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Relu_740_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_742<<<dim3(2, 16, 2), dim3(16, 2, 8), 0, 0>>>(DepthwiseConv2dNative_736_0, Constant_2173_0, Convolution_742_0);
// Convolution_float_float_float_cuda_Convolution_744<<<dim3(2, 16, 2), dim3(16, 2, 8), 0, 0>>>(DepthwiseConv2dNative_737_0, Constant_2176_0, Convolution_744_0);
// Convolution_float_float_float_cuda_Convolution_746<<<dim3(2, 16, 2), dim3(16, 2, 8), 0, 0>>>(DepthwiseConv2dNative_738_0, Constant_2179_0, Convolution_746_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3021<<<dim3(2, 16, 2), dim3(16, 2, 8), 0, 0>>>(DepthwiseConv2dNative_721_0, Constant_2167_0, Constant_3022_0, Relu_739_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3023<<<dim3(2, 16, 2), dim3(16, 2, 8), 0, 0>>>(DepthwiseConv2dNative_722_0, Constant_2170_0, Constant_3024_0, Relu_740_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Convolution_float_float_float_cuda_Convolution_744 : Convolution_float_float_float_cuda_Convolution_742
// Convolution_float_float_float_cuda_Convolution_746 : Convolution_float_float_float_cuda_Convolution_742
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3023 : Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3021

// Node name:	Convolution_742
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_736_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2173_0	type: float	shape: Shape{32, 32, 1, 1}
// Output:
//	- name: Convolution_742_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_742_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(16, 2, 8);
    const dim3 gridDim(2, 16, 2);
    const dim3 threadIdx(thread_id % 16, thread_id / 16 % 2, thread_id / 32);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 16, block_id / 32);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 2048);
    {
        float* compute = output0;{
           float compute_local[2];
          
          
          for (int ff_c_init = 0; ff_c_init < 2; ++ff_c_init) {
            compute_local[ff_c_init] = 0.000000e+00f;
          }
          for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
            __syncthreads();
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
              pad_temp_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] = input0[(((((((rc_outer * 16384) + (((int)threadIdx.z) * 2048)) + (((int)threadIdx.y) * 1024)) + (((int)blockIdx.y) * 64)) + ((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 4) * 32)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 15))];
            }
            input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 32)) + (rc_outer * 16)) + ((int)threadIdx.x))];
            __syncthreads();
            for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
              for (int ff_c = 0; ff_c < 2; ++ff_c) {
                compute_local[ff_c] = (compute_local[ff_c] + (pad_temp_shared[(((rc_inner * 32) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x))] * input1_shared[(((((int)threadIdx.z) * 32) + (ff_c * 16)) + rc_inner)]));
              }
            }
          }
          for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 2; ++ff_inner_inner_inner) {
            compute[(((((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 2048)) + (ff_inner_inner_inner * 1024)) + (((int)blockIdx.y) * 64)) + (((int)threadIdx.y) * 32)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x))] = compute_local[ff_inner_inner_inner];
          }
        }


    }

}
// Node name:	Matched_Pattern_3021
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_721_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2167_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: Constant_3022_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Relu_739_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3021_block_kernel(float* input0, float* input1, float* input2, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(16, 2, 8);
    const dim3 gridDim(2, 16, 2);
    const dim3 threadIdx(thread_id % 16, thread_id / 16 % 2, thread_id / 32);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 16, block_id / 32);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 2048);
    {
        float* compute = output0;{
           float compute1[2];
          
          
          for (int ff_init = 0; ff_init < 2; ++ff_init) {
            compute1[ff_init] = 0.000000e+00f;
          }
          for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
            __syncthreads();
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
              pad_temp_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] = input0[(((((((rc_outer * 16384) + (((int)threadIdx.z) * 2048)) + (((int)threadIdx.y) * 1024)) + (((int)blockIdx.y) * 64)) + ((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 4) * 32)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 15))];
            }
            input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 32)) + (rc_outer * 16)) + ((int)threadIdx.x))];
            __syncthreads();
            for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
              for (int ff = 0; ff < 2; ++ff) {
                compute1[ff] = (compute1[ff] + (pad_temp_shared[(((rc_inner * 32) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x))] * input1_shared[(((((int)threadIdx.z) * 32) + (ff * 16)) + rc_inner)]));
              }
            }
          }
          for (int i1_inner_inner_inner = 0; i1_inner_inner_inner < 2; ++i1_inner_inner_inner) {
            compute[(((((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 2048)) + (i1_inner_inner_inner * 1024)) + (((int)blockIdx.y) * 64)) + (((int)threadIdx.y) * 32)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x))] = max((compute1[i1_inner_inner_inner] + input2[(((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 2)) + i1_inner_inner_inner)]), 0.000000e+00f);
          }
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Convolution_Convolution_Convolution_Matched_Pattern_Matched_Pattern_32(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* input11, float* output0, float* output1, float* output2, float* output3, float* output4)
{
    __shared__ char shared_buffer[3072];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Convolution_float_float_float_cuda_Convolution_742_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Convolution_float_float_float_cuda_Convolution_742_block_kernel(input2, input3, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191)
    {
        Convolution_float_float_float_cuda_Convolution_742_block_kernel(input4, input5, output2, threadIdx.x, blockIdx.x - 128 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 192 && (int)blockIdx.x <= 255)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3021_block_kernel(input6, input7, input8, output3, threadIdx.x, blockIdx.x - 192 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 256 && (int)blockIdx.x <= 319)
    {
        Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3021_block_kernel(input9, input10, input11, output4, threadIdx.x, blockIdx.x - 256 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Convolution_Convolution_Convolution_Matched_Pattern_Matched_Pattern_32_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* input11, float* output0, float* output1, float* output2, float* output3, float* output4) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Convolution_Convolution_Convolution_Matched_Pattern_Matched_Pattern_32<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, input6, input7, input8, input9, input10, input11, output0, output1, output2, output3, output4);
}
// Node name:	 BlockFusion
// Input:
//	- name: Convolution_577_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2760_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2827_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_575_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: BatchNormInference_579_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: BatchNormInference_578_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Relu_580_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Add_float_float_float_cuda_Add_2082<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_577_0, Constant_2760_0, BatchNormInference_579_0);
// FusedKernel_float_float_float_float_cuda_Add_Relu_7<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_575_0, Constant_2827_0, Relu_580_0, BatchNormInference_578_0);
// Deduped function map: <src_function_name : deduped_function_name>

// Node name:	Add_2082
// Description:	Add
// Input:
//	- name: Convolution_577_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2760_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: BatchNormInference_579_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void Add_float_float_float_cuda_Add_2082_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = add(input0[blockIdx.x * 512 + threadIdx.x], input1[blockIdx.x * 512 + threadIdx.x]);

}
// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_575_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2827_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Relu_580_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: BatchNormInference_578_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Add_float_float_float_cuda_Add_2085<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_575_0, Constant_2827_0, BatchNormInference_578_0);
// Relu_float_float_cuda_Relu_580<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(BatchNormInference_578_0, Relu_580_0);
__device__ __noinline__ void FusedKernel_float_float_float_float_cuda_Add_Relu_7_block_kernel(float* input0, float* input1, float* output0, float* output1, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    int tid = blockIdx.x * 512 + threadIdx.x;
    float temp0 = add(input0[tid], input1[tid]);
    float temp1 = relu(temp0);
    output1[tid] = temp0;
    output0[tid] = temp1;

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_cuda_Add_fused_kernel_10(float* input0, float* input1, float* input2, float* input3, float* output0, float* output1, float* output2)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Add_float_float_float_cuda_Add_2082_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        FusedKernel_float_float_float_float_cuda_Add_Relu_7_block_kernel(input3, input2, output2, output1, threadIdx.x, blockIdx.x - 64 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_cuda_Add_fused_kernel_10_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* output0, float* output1, float* output2) {
    BlockFusionKernel_float_float_float_float_float_float_float_cuda_Add_fused_kernel_10<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0, output1, output2);
}
