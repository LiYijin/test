#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float relu(float x0)
{
    return fmaxf(0,x0);
}
__device__ __forceinline__ float add(float x0, float x1)
{
    return x0 + x1;
}
// Node name:	Constant_142
// Description:	Constant
// Input:
// Output:
//	- name: Constant_142_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_142(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_142_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_142_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1152];
    bin_file.read(tmp_mem, 1152);
    hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_3134
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3134_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_3134(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_3134_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_3134_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_450
// Description:	Constant
// Input:
// Output:
//	- name: Constant_450_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_450(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_450_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_450_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[2304];
    bin_file.read(tmp_mem, 2304);
    hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_263
// Description:	Constant
// Input:
// Output:
//	- name: Constant_263_0	type: float	shape: Shape{5, 5, 32, 1}
void Constant_float_cuda_Constant_263(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_263_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_263_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[3200];
    bin_file.read(tmp_mem, 3200);
    hipMemcpyAsync(output0, tmp_mem, 3200, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2937
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2937_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2937(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2937_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2937_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2774
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2774_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2774(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2774_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2774_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2440
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2440_0	type: float	shape: Shape{64, 384, 1, 1}
void Constant_float_cuda_Constant_2440(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2440_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2440_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[98304];
    bin_file.read(tmp_mem, 98304);
    hipMemcpyAsync(output0, tmp_mem, 98304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2413
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2413_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2413(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2413_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2413_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2842
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2842_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2842(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2842_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2842_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[131072];
    bin_file.read(tmp_mem, 131072);
    hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2332
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2332_0	type: float	shape: Shape{64, 384, 1, 1}
void Constant_float_cuda_Constant_2332(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2332_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2332_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[98304];
    bin_file.read(tmp_mem, 98304);
    hipMemcpyAsync(output0, tmp_mem, 98304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_1743_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_255_0	type: float	shape: Shape{5, 5, 128, 1}
//	- name: Relu_1744_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_119_0	type: float	shape: Shape{3, 3, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1745_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1746_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1745<<<dim3(64, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1743_0, Constant_255_0, DepthwiseConv2dNative_1745_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1746<<<dim3(64, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1744_0, Constant_119_0, DepthwiseConv2dNative_1746_0);
// Deduped function map: <src_function_name : deduped_function_name>

// Node name:	DepthwiseConv2dNative_1745
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1743_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_255_0	type: float	shape: Shape{5, 5, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1745_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1745_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 8;
        const int in_width = 8;
        const int in_depth = 128;
        const int filter_height = 5;
        const int filter_width = 5;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 2;
        const int pad_width = 2;
        const int out_height = 8;
        const int out_width = 8;
        const int out_depth = 128;
        const int num_outputs = 8192;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
// Node name:	DepthwiseConv2dNative_1746
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1744_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_119_0	type: float	shape: Shape{3, 3, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1746_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1746_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 8;
        const int in_width = 8;
        const int in_depth = 128;
        const int filter_height = 3;
        const int filter_width = 3;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 1;
        const int pad_width = 1;
        const int out_height = 8;
        const int out_width = 8;
        const int out_depth = 128;
        const int num_outputs = 8192;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_177(float* input0, float* input1, float* input2, float* input3, float* output0, float* output1)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1745_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1746_block_kernel(input2, input3, output1, threadIdx.x, blockIdx.x - 64 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_177_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_177<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0, output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_1250_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_164_0	type: float	shape: Shape{3, 3, 64, 1}
//	- name: Relu_1251_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_403_0	type: float	shape: Shape{5, 5, 64, 1}
//	- name: Convolution_1262_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2925_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2801_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1258_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2923_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1260_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: DepthwiseConv2dNative_1255_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1256_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: BatchNormInference_1269_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Add_1272_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1255<<<dim3(128, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1250_0, Constant_164_0, DepthwiseConv2dNative_1255_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1256<<<dim3(128, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1251_0, Constant_403_0, DepthwiseConv2dNative_1256_0);
// Add_float_float_float_cuda_Add_2466<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1262_0, Constant_2925_0, BatchNormInference_1269_0);
// FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_45<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1258_0, Constant_2801_0, Convolution_1260_0, Constant_2923_0, Add_1272_0);
// Deduped function map: <src_function_name : deduped_function_name>

// Node name:	DepthwiseConv2dNative_1255
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1250_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_164_0	type: float	shape: Shape{3, 3, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_1255_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1255_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(128, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 16;
        const int in_width = 16;
        const int in_depth = 64;
        const int filter_height = 3;
        const int filter_width = 3;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 1;
        const int pad_width = 1;
        const int out_height = 16;
        const int out_width = 16;
        const int out_depth = 64;
        const int num_outputs = 16384;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
// Node name:	DepthwiseConv2dNative_1256
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1251_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_403_0	type: float	shape: Shape{5, 5, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_1256_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1256_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(128, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 16;
        const int in_width = 16;
        const int in_depth = 64;
        const int filter_height = 5;
        const int filter_width = 5;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 2;
        const int pad_width = 2;
        const int out_height = 16;
        const int out_width = 16;
        const int out_depth = 64;
        const int num_outputs = 16384;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
// Node name:	Add_2466
// Description:	Add
// Input:
//	- name: Convolution_1262_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2925_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: BatchNormInference_1269_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Add_float_float_float_cuda_Add_2466_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(32, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = add(input0[blockIdx.x * 512 + threadIdx.x], input1[blockIdx.x * 512 + threadIdx.x]);

}
// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_1258_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2801_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1260_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2923_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Add_1272_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Add_float_float_float_cuda_Add_2460<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1258_0, Constant_2801_0, BatchNormInference_1267_0);
// Add_float_float_float_cuda_Add_2463<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1260_0, Constant_2923_0, BatchNormInference_1268_0);
// Add_float_float_float_cuda_Add_1272<<<dim3(32, 1, 1), dim3(512, 1, 1), 0, 0>>>(BatchNormInference_1268_0, BatchNormInference_1267_0, Add_1272_0);
__device__ __noinline__ void FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_45_block_kernel(float* input0, float* input1, float* input2, float* input3, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(32, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    int tid = blockIdx.x * 512 + threadIdx.x;
    float temp0 = add(input0[tid], input1[tid]);
    float temp1 = add(input2[tid], input3[tid]);
    float temp2 = add(temp1, temp0);
    output0[tid] = temp2;

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_fused_kernel_107(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* output0, float* output1, float* output2, float* output3)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 127)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1255_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 255)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1256_block_kernel(input2, input3, output1, threadIdx.x, blockIdx.x - 128 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 256 && (int)blockIdx.x <= 287)
    {
        Add_float_float_float_cuda_Add_2466_block_kernel(input4, input5, output2, threadIdx.x, blockIdx.x - 256 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 288 && (int)blockIdx.x <= 319)
    {
        FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_45_block_kernel(input7, input6, input9, input8, output3, threadIdx.x, blockIdx.x - 288 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_fused_kernel_107_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* output0, float* output1, float* output2, float* output3) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_fused_kernel_107<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, input6, input7, input8, input9, output0, output1, output2, output3);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_1129_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2395_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: DepthwiseConv2dNative_1130_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2398_0	type: float	shape: Shape{64, 64, 1, 1}
// Output:
//	- name: Convolution_1138_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1140_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_1138<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1129_0, Constant_2395_0, Convolution_1138_0);
// Convolution_float_float_float_cuda_Convolution_1140<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_1130_0, Constant_2398_0, Convolution_1140_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Convolution_float_float_float_cuda_Convolution_1140 : Convolution_float_float_float_cuda_Convolution_1138

// Node name:	Convolution_1138
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_1129_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2395_0	type: float	shape: Shape{64, 64, 1, 1}
// Output:
//	- name: Convolution_1138_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_1138_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(8, 2, 16);
    const dim3 gridDim(2, 8, 4);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 8, block_id / 16);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 1024);
    {
        float* compute = output0;{
           float compute_local[1];
          
          
          compute_local[0] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[(((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 4096)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 16)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 8192)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 32)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 12288)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 48)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          compute[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))] = compute_local[0];
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_90(float* input0, float* input1, float* input2, float* input3, float* output0, float* output1)
{
    __shared__ char shared_buffer[2048];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Convolution_float_float_float_cuda_Convolution_1138_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Convolution_float_float_float_cuda_Convolution_1138_block_kernel(input2, input3, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_90_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_90<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0, output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_1580_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2644_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: DepthwiseConv2dNative_1581_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2647_0	type: float	shape: Shape{128, 128, 1, 1}
// Output:
//	- name: Convolution_1589_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1591_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_1589<<<dim3(1, 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1580_0, Constant_2644_0, Convolution_1589_0);
// Convolution_float_float_float_cuda_Convolution_1591<<<dim3(1, 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1581_0, Constant_2647_0, Convolution_1591_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Convolution_float_float_float_cuda_Convolution_1591 : Convolution_float_float_float_cuda_Convolution_1589

// Node name:	Convolution_1589
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_1580_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2644_0	type: float	shape: Shape{128, 128, 1, 1}
// Output:
//	- name: Convolution_1589_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_1589_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(8, 2, 8);
    const dim3 gridDim(1, 4, 16);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 1, block_id / 1 % 4, block_id / 4);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 1024);
    {
        float* compute = output0;{
           float compute_local[1];
          
          
          compute_local[0] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2))];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1024)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1025)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 16)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2048)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2049)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 32)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 3072)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 3073)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 48)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 4096)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 4097)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 64)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 5120)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 5121)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 80)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 6144)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 6145)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 96)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 7168)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 7169)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 112)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          compute[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = compute_local[0];
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_155(float* input0, float* input1, float* input2, float* input3, float* output0, float* output1)
{
    __shared__ char shared_buffer[1536];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Convolution_float_float_float_cuda_Convolution_1589_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Convolution_float_float_float_cuda_Convolution_1589_block_kernel(input2, input3, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_155_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_155<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0, output1);
}
// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_466_0	type: float	shape: Shape{1, 96, 32, 32}
//	- name: Constant_2815_0	type: float	shape: Shape{1, 96, 32, 32}
// Output:
//	- name: Relu_468_0	type: float	shape: Shape{1, 96, 32, 32}
//	- name: BatchNormInference_467_0	type: float	shape: Shape{1, 96, 32, 32}
// Fused functions:
// Add_float_float_float_cuda_Add_2013<<<dim3(192, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_466_0, Constant_2815_0, BatchNormInference_467_0);
// Relu_float_float_cuda_Relu_468<<<dim3(192, 1, 1), dim3(512, 1, 1), 0, 0>>>(BatchNormInference_467_0, Relu_468_0);
extern "C" __launch_bounds__(512) __global__ void FusedKernel_float_float_float_float_cuda_Add_Relu_0(float* input0, float* input1, float* output0, float* output1)
{
    int tid = blockIdx.x * 512 + threadIdx.x;
    float temp0 = add(input0[tid], input1[tid]);
    float temp1 = relu(temp0);
    output1[tid] = temp0;
    output0[tid] = temp1;

}
extern void FusedKernel_float_float_float_float_cuda_Add_Relu_0_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* output0, float* output1) {
    FusedKernel_float_float_float_float_cuda_Add_Relu_0<<<grids, blocks, mem, stream>>>(input0, input1, output0, output1);
}
