#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1)
{
    return x0 + x1;
}
// Node name:	Constant_329
// Description:	Constant
// Input:
// Output:
//	- name: Constant_329_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_329(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_329_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_329_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[2304];
    bin_file.read(tmp_mem, 2304);
    hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2488
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2488_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2488(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2488_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2488_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2802
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2802_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2802(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2802_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2802_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2350
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2350_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2350(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2350_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2350_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2149
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2149_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2149(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2149_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2149_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4096];
    bin_file.read(tmp_mem, 4096);
    hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_167
// Description:	Constant
// Input:
// Output:
//	- name: Constant_167_0	type: float	shape: Shape{5, 5, 64, 1}
void Constant_float_cuda_Constant_167(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_167_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_167_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[6400];
    bin_file.read(tmp_mem, 6400);
    hipMemcpyAsync(output0, tmp_mem, 6400, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2266
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2266_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2266(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2266_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2266_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2977
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2977_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2977(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2977_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2977_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2281
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2281_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2281(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2281_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2281_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2236
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2236_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2236(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2236_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2236_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2560
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2560_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2560(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2560_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2560_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2053
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2053_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2053(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2053_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2053_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4096];
    bin_file.read(tmp_mem, 4096);
    hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	 BlockFusion
// Input:
//	- name: BatchNormInference_1092_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_1095_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_425_0	type: float	shape: Shape{3, 3, 64, 1}
//	- name: Constant_291_0	type: float	shape: Shape{5, 5, 64, 1}
//	- name: Constant_282_0	type: float	shape: Shape{3, 3, 64, 1}
// Output:
//	- name: Slice_1094_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1101_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1099_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1100_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Slice_float_float_cuda_Slice_1094<<<dim3(256, 1, 1), dim3(64, 1, 1), 0, 0>>>(BatchNormInference_1092_0, Slice_1094_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1101<<<dim3(128, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1095_0, Constant_425_0, DepthwiseConv2dNative_1101_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1099<<<dim3(128, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1095_0, Constant_291_0, DepthwiseConv2dNative_1099_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1100<<<dim3(128, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1095_0, Constant_282_0, DepthwiseConv2dNative_1100_0);
// Deduped function map: <src_function_name : deduped_function_name>
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1100 : DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1101

// Node name:	Slice_1094
// Description:	Slice
// Input:
//	- name: BatchNormInference_1092_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Slice_1094_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Slice_float_float_cuda_Slice_1094_block_kernel(float* input0, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 64){
        return;
    }
    const dim3 blockDim(64, 1, 1);
    const dim3 gridDim(256, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < 16384)
    {
        uint32_t input_strides[] = {16384, 256, 16, 1};
        uint32_t output_strides[] = {16384, 256, 16, 1};
        uint32_t lower_bounds[] = {0, 0, 0, 0};
        uint32_t slice_strides[] = {1, 1, 1, 1};
        uint32_t input_idx = 0;
        uint32_t output_idx = tid;
        input_idx += (((output_idx / output_strides[0]) * slice_strides[0]) + lower_bounds[0]) * input_strides[0];
        output_idx %= output_strides[0];
        input_idx += (((output_idx / output_strides[1]) * slice_strides[1]) + lower_bounds[1]) * input_strides[1];
        output_idx %= output_strides[1];
        input_idx += (((output_idx / output_strides[2]) * slice_strides[2]) + lower_bounds[2]) * input_strides[2];
        output_idx %= output_strides[2];
        input_idx += (((output_idx / output_strides[3]) * slice_strides[3]) + lower_bounds[3]) * input_strides[3];
        output0[tid] = input0[input_idx];
    }

}
// Node name:	DepthwiseConv2dNative_1101
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1095_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_425_0	type: float	shape: Shape{3, 3, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_1101_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1101_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(128, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 16;
        const int in_width = 16;
        const int in_depth = 64;
        const int filter_height = 3;
        const int filter_width = 3;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 1;
        const int pad_width = 1;
        const int out_height = 16;
        const int out_width = 16;
        const int out_depth = 64;
        const int num_outputs = 16384;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
// Node name:	DepthwiseConv2dNative_1099
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1095_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_291_0	type: float	shape: Shape{5, 5, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_1099_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1099_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(128, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 16;
        const int in_width = 16;
        const int in_depth = 64;
        const int filter_height = 5;
        const int filter_width = 5;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 2;
        const int pad_width = 2;
        const int out_height = 16;
        const int out_width = 16;
        const int out_depth = 64;
        const int num_outputs = 16384;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_85(float* input0, float* input1, float* input2, float* input3, float* input4, float* output0, float* output1, float* output2, float* output3)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 255)
    {
        Slice_float_float_cuda_Slice_1094_block_kernel(input0, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 256 && (int)blockIdx.x <= 383)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1101_block_kernel(input1, input2, output1, threadIdx.x, blockIdx.x - 256 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 384 && (int)blockIdx.x <= 511)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1099_block_kernel(input1, input3, output2, threadIdx.x, blockIdx.x - 384 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 512 && (int)blockIdx.x <= 639)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1101_block_kernel(input1, input4, output3, threadIdx.x, blockIdx.x - 512 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_85_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* output0, float* output1, float* output2, float* output3) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_85<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, output0, output1, output2, output3);
}
// Node name:	 BlockFusion
// Input:
//	- name: Constant_2958_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1589_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Slice_1545_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2959_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1591_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1525_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1598_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Add_1599_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// FusedKernel_float_float_float_float_cuda_Add_Add_63<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1589_0, Constant_2958_0, Slice_1545_0, Add_1598_0);
// FusedKernel_float_float_float_float_cuda_Add_Add_64<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1591_0, Constant_2959_0, BatchNormInference_1525_0, Add_1599_0);
// Deduped function map: <src_function_name : deduped_function_name>
// FusedKernel_float_float_float_float_cuda_Add_Add_64 : FusedKernel_float_float_float_float_cuda_Add_Add_63

// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_1589_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2958_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Slice_1545_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1598_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Add_float_float_float_cuda_Add_2646<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1589_0, Constant_2958_0, BatchNormInference_1595_0);
// Add_float_float_float_cuda_Add_1598<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(BatchNormInference_1595_0, Slice_1545_0, Add_1598_0);
__device__ __noinline__ void FusedKernel_float_float_float_float_cuda_Add_Add_63_block_kernel(float* input0, float* input1, float* input2, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(16, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    int tid = blockIdx.x * 512 + threadIdx.x;
    float temp0 = add(input0[tid], input1[tid]);
    float temp1 = add(temp0, input2[tid]);
    output0[tid] = temp1;

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_156(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* output0, float* output1)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15)
    {
        FusedKernel_float_float_float_float_cuda_Add_Add_63_block_kernel(input1, input0, input2, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31)
    {
        FusedKernel_float_float_float_float_cuda_Add_Add_63_block_kernel(input4, input3, input5, output1, threadIdx.x, blockIdx.x - 16 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_156_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_156<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, output0, output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: Constant_2743_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1520_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1522_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2745_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1518_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2951_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1515_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_40_0	type: float	shape: Shape{5, 5, 128, 1}
//	- name: Relu_1516_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_146_0	type: float	shape: Shape{3, 3, 128, 1}
// Output:
//	- name: Add_1532_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1525_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1523_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1524_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_58<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1520_0, Constant_2743_0, Convolution_1522_0, Constant_2745_0, Add_1532_0);
// Add_float_float_float_cuda_Add_2601<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1518_0, Constant_2951_0, BatchNormInference_1525_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1523<<<dim3(64, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1515_0, Constant_40_0, DepthwiseConv2dNative_1523_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1524<<<dim3(64, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1516_0, Constant_146_0, DepthwiseConv2dNative_1524_0);
// Deduped function map: <src_function_name : deduped_function_name>

// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_1520_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2743_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1522_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2745_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1532_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Add_float_float_float_cuda_Add_2604<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1520_0, Constant_2743_0, BatchNormInference_1526_0);
// Add_float_float_float_cuda_Add_2607<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1522_0, Constant_2745_0, BatchNormInference_1527_0);
// Add_float_float_float_cuda_Add_1532<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(BatchNormInference_1526_0, BatchNormInference_1527_0, Add_1532_0);
__device__ __noinline__ void FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_58_block_kernel(float* input0, float* input1, float* input2, float* input3, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(16, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    int tid = blockIdx.x * 512 + threadIdx.x;
    float temp0 = add(input0[tid], input1[tid]);
    float temp1 = add(input2[tid], input3[tid]);
    float temp2 = add(temp0, temp1);
    output0[tid] = temp2;

}
// Node name:	Add_2601
// Description:	Add
// Input:
//	- name: Convolution_1518_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2951_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: BatchNormInference_1525_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void Add_float_float_float_cuda_Add_2601_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(16, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = add(input0[blockIdx.x * 512 + threadIdx.x], input1[blockIdx.x * 512 + threadIdx.x]);

}
// Node name:	DepthwiseConv2dNative_1523
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1515_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_40_0	type: float	shape: Shape{5, 5, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1523_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1523_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 8;
        const int in_width = 8;
        const int in_depth = 128;
        const int filter_height = 5;
        const int filter_width = 5;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 2;
        const int pad_width = 2;
        const int out_height = 8;
        const int out_width = 8;
        const int out_depth = 128;
        const int num_outputs = 8192;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
// Node name:	DepthwiseConv2dNative_1524
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1516_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_146_0	type: float	shape: Shape{3, 3, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1524_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1524_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 8;
        const int in_width = 8;
        const int in_depth = 128;
        const int filter_height = 3;
        const int filter_width = 3;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 1;
        const int pad_width = 1;
        const int out_height = 8;
        const int out_width = 8;
        const int out_depth = 128;
        const int num_outputs = 8192;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_fused_kernel_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_145(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* output0, float* output1, float* output2, float* output3)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15)
    {
        FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_58_block_kernel(input1, input0, input2, input3, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31)
    {
        Add_float_float_float_cuda_Add_2601_block_kernel(input4, input5, output1, threadIdx.x, blockIdx.x - 16 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 95)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1523_block_kernel(input6, input7, output2, threadIdx.x, blockIdx.x - 32 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 96 && (int)blockIdx.x <= 159)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1524_block_kernel(input8, input9, output3, threadIdx.x, blockIdx.x - 96 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_fused_kernel_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_145_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* output0, float* output1, float* output2, float* output3) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_fused_kernel_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_145<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, input6, input7, input8, input9, output0, output1, output2, output3);
}
