#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1)
{
    return x0 + x1;
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_862_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2251_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: DepthwiseConv2dNative_860_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2245_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: DepthwiseConv2dNative_861_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2248_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: AvgPool_837_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: BatchNormInference_768_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Relu_838_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_121_0	type: float	shape: Shape{5, 5, 32, 1}
//	- name: Constant_184_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: Convolution_870_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_866_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_868_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Add_845_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_846_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_847_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_870<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_862_0, Constant_2251_0, Convolution_870_0);
// Convolution_float_float_float_cuda_Convolution_866<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_860_0, Constant_2245_0, Convolution_866_0);
// Convolution_float_float_float_cuda_Convolution_868<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(DepthwiseConv2dNative_861_0, Constant_2248_0, Convolution_868_0);
// Add_float_float_float_cuda_Add_845<<<dim3(64, 1, 1), dim3(512, 1, 1), 0, 0>>>(AvgPool_837_0, BatchNormInference_768_0, Add_845_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_846<<<dim3(256, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_838_0, Constant_121_0, DepthwiseConv2dNative_846_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_847<<<dim3(256, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_838_0, Constant_184_0, DepthwiseConv2dNative_847_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Convolution_float_float_float_cuda_Convolution_866 : Convolution_float_float_float_cuda_Convolution_870
// Convolution_float_float_float_cuda_Convolution_868 : Convolution_float_float_float_cuda_Convolution_870

// Node name:	Convolution_870
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_862_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2251_0	type: float	shape: Shape{64, 64, 1, 1}
// Output:
//	- name: Convolution_870_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_870_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(8, 2, 16);
    const dim3 gridDim(2, 8, 4);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 8, block_id / 16);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 1024);
    {
        float* compute = output0;{
           float compute_local[1];
          
          
          compute_local[0] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[(((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 4096)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 16)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 8192)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 32)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input0[((((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 12288)];
          input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) + 48)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 16)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 16) + 1)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 16) + 2)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 16) + 3)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 16) + 4)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 16) + 5)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 16) + 6)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 16) + 7)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 16) + 8)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 16) + 9)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 16) + 10)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 16) + 11)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 16) + 12)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 16) + 13)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 16) + 14)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 16) + 15)]));
          compute[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))] = compute_local[0];
        }


    }

}
// Node name:	Add_845
// Description:	Add
// Input:
//	- name: AvgPool_837_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: BatchNormInference_768_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Add_845_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void Add_float_float_float_cuda_Add_845_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(64, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    output0[blockIdx.x * 512 + threadIdx.x] = add(input0[blockIdx.x * 512 + threadIdx.x], input1[blockIdx.x * 512 + threadIdx.x]);

}
// Node name:	DepthwiseConv2dNative_846
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_838_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_121_0	type: float	shape: Shape{5, 5, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_846_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_846_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(256, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 32;
        const int in_width = 32;
        const int in_depth = 32;
        const int filter_height = 5;
        const int filter_width = 5;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 2;
        const int pad_width = 2;
        const int out_height = 32;
        const int out_width = 32;
        const int out_depth = 32;
        const int num_outputs = 32768;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
// Node name:	DepthwiseConv2dNative_847
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_838_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_184_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_847_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_847_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(256, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 32;
        const int in_width = 32;
        const int in_depth = 32;
        const int filter_height = 3;
        const int filter_width = 3;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 1;
        const int pad_width = 1;
        const int out_height = 32;
        const int out_width = 32;
        const int out_depth = 32;
        const int num_outputs = 32768;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Convolution_Convolution_Convolution_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_49(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* output0, float* output1, float* output2, float* output3, float* output4, float* output5)
{
    __shared__ char shared_buffer[2048];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Convolution_float_float_float_cuda_Convolution_870_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Convolution_float_float_float_cuda_Convolution_870_block_kernel(input2, input3, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191)
    {
        Convolution_float_float_float_cuda_Convolution_870_block_kernel(input4, input5, output2, threadIdx.x, blockIdx.x - 128 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 192 && (int)blockIdx.x <= 255)
    {
        Add_float_float_float_cuda_Add_845_block_kernel(input6, input7, output3, threadIdx.x, blockIdx.x - 192 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 256 && (int)blockIdx.x <= 511)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_846_block_kernel(input8, input9, output4, threadIdx.x, blockIdx.x - 256 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 512 && (int)blockIdx.x <= 767)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_847_block_kernel(input8, input10, output5, threadIdx.x, blockIdx.x - 512 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Convolution_Convolution_Convolution_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_49_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* input9, float* input10, float* output0, float* output1, float* output2, float* output3, float* output4, float* output5) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Convolution_Convolution_Convolution_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_49<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, input6, input7, input8, input9, input10, output0, output1, output2, output3, output4, output5);
}
// Node name:	Constant_2353
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2353_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2353(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2353_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2353_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_413
// Description:	Constant
// Input:
// Output:
//	- name: Constant_413_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_413(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_413_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_413_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4608];
    bin_file.read(tmp_mem, 4608);
    hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_451
// Description:	Constant
// Input:
// Output:
//	- name: Constant_451_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_451(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_451_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_451_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1152];
    bin_file.read(tmp_mem, 1152);
    hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2743
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2743_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2743(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2743_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2743_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2209
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2209_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2209(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2209_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2209_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4096];
    bin_file.read(tmp_mem, 4096);
    hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_3088
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3088_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_3088(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_3088_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_3088_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2455
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2455_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2455(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2455_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2455_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_27
// Description:	Constant
// Input:
// Output:
//	- name: Constant_27_0	type: float	shape: Shape{5, 5, 64, 1}
void Constant_float_cuda_Constant_27(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_27_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_27_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[6400];
    bin_file.read(tmp_mem, 6400);
    hipMemcpyAsync(output0, tmp_mem, 6400, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2668
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2668_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2668(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2668_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2668_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_63
// Description:	Constant
// Input:
// Output:
//	- name: Constant_63_0	type: float	shape: Shape{5, 5, 128, 1}
void Constant_float_cuda_Constant_63(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_63_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_63_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[12800];
    bin_file.read(tmp_mem, 12800);
    hipMemcpyAsync(output0, tmp_mem, 12800, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_153
// Description:	Constant
// Input:
// Output:
//	- name: Constant_153_0	type: float	shape: Shape{5, 5, 64, 1}
void Constant_float_cuda_Constant_153(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_153_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_153_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[6400];
    bin_file.read(tmp_mem, 6400);
    hipMemcpyAsync(output0, tmp_mem, 6400, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	 BlockFusion
// Input:
//	- name: BatchNormInference_467_0	type: float	shape: Shape{1, 96, 32, 32}
//	- name: Constant_2014_0	type: float	shape: Shape{32, 96, 1, 1}
//	- name: Relu_468_0	type: float	shape: Shape{1, 96, 32, 32}
//	- name: Constant_2029_0	type: float	shape: Shape{32, 96, 1, 1}
//	- name: Constant_73_0	type: float	shape: Shape{3, 3, 96, 1}
//	- name: Constant_2026_0	type: float	shape: Shape{32, 96, 1, 1}
//	- name: Constant_423_0	type: float	shape: Shape{5, 5, 96, 1}
//	- name: Constant_69_0	type: float	shape: Shape{3, 3, 96, 1}
//	- name: Constant_2023_0	type: float	shape: Shape{32, 96, 1, 1}
// Output:
//	- name: Convolution_471_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_474_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_472_0	type: float	shape: Shape{1, 96, 32, 32}
//	- name: Convolution_480_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_478_0	type: float	shape: Shape{1, 96, 32, 32}
//	- name: DepthwiseConv2dNative_477_0	type: float	shape: Shape{1, 96, 32, 32}
//	- name: Convolution_476_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_471<<<dim3(1, 32, 2), dim3(32, 1, 8), 0, 0>>>(BatchNormInference_467_0, Constant_2014_0, Convolution_471_0);
// Convolution_float_float_float_cuda_Convolution_474<<<dim3(1, 32, 2), dim3(32, 1, 8), 0, 0>>>(Relu_468_0, Constant_2029_0, Convolution_474_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_472<<<dim3(768, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_468_0, Constant_73_0, DepthwiseConv2dNative_472_0);
// Convolution_float_float_float_cuda_Convolution_480<<<dim3(1, 32, 2), dim3(32, 1, 8), 0, 0>>>(Relu_468_0, Constant_2026_0, Convolution_480_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_478<<<dim3(768, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_468_0, Constant_423_0, DepthwiseConv2dNative_478_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_477<<<dim3(768, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_468_0, Constant_69_0, DepthwiseConv2dNative_477_0);
// Convolution_float_float_float_cuda_Convolution_476<<<dim3(1, 32, 2), dim3(32, 1, 8), 0, 0>>>(Relu_468_0, Constant_2023_0, Convolution_476_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Convolution_float_float_float_cuda_Convolution_474 : Convolution_float_float_float_cuda_Convolution_471
// Convolution_float_float_float_cuda_Convolution_480 : Convolution_float_float_float_cuda_Convolution_471
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_477 : DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_472
// Convolution_float_float_float_cuda_Convolution_476 : Convolution_float_float_float_cuda_Convolution_471

// Node name:	Convolution_471
// Description:	Convolution
// Input:
//	- name: BatchNormInference_467_0	type: float	shape: Shape{1, 96, 32, 32}
//	- name: Constant_2014_0	type: float	shape: Shape{32, 96, 1, 1}
// Output:
//	- name: Convolution_471_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void Convolution_float_float_float_cuda_Convolution_471_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 blockDim(32, 1, 8);
    const dim3 gridDim(1, 32, 2);
    const dim3 threadIdx(thread_id % 32, 0, thread_id / 32);
    const dim3 blockIdx(block_id % 1, block_id / 1 % 32, block_id / 32);
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 2048);
    {
        float* compute = output0;{
           float compute_local[2];
          
          
          compute_local[0] = 0.000000e+00f;
          compute_local[1] = 0.000000e+00f;
          #pragma unroll
          for (int rc_outer = 0; rc_outer < 6; ++rc_outer) {
            __syncthreads();
            #pragma unroll
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
              pad_temp_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] = input0[(((((rc_outer * 16384) + (((int)threadIdx.z) * 2048)) + ((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 5) * 1024)) + (((int)blockIdx.y) * 32)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 31))];
            }
            input1_shared[((((int)threadIdx.z) * 32) + ((int)threadIdx.x))] = input1[(((((((int)blockIdx.z) * 1536) + (((int)threadIdx.z) * 192)) + ((((int)threadIdx.x) >> 4) * 96)) + (rc_outer * 16)) + (((int)threadIdx.x) & 15))];
            __syncthreads();
            #pragma unroll
            for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
              compute_local[0] = (compute_local[0] + (pad_temp_shared[((rc_inner * 32) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 16) + rc_inner)]));
              compute_local[1] = (compute_local[1] + (pad_temp_shared[((rc_inner * 32) + ((int)threadIdx.x))] * input1_shared[(((((int)threadIdx.z) * 16) + rc_inner) + 128)]));
            }
          }
          compute[((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 1024)) + (((int)blockIdx.y) * 32)) + ((int)threadIdx.x))] = compute_local[0];
          compute[(((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 1024)) + (((int)blockIdx.y) * 32)) + ((int)threadIdx.x)) + 8192)] = compute_local[1];
        }


    }

}
// Node name:	DepthwiseConv2dNative_472
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_468_0	type: float	shape: Shape{1, 96, 32, 32}
//	- name: Constant_73_0	type: float	shape: Shape{3, 3, 96, 1}
// Output:
//	- name: DepthwiseConv2dNative_472_0	type: float	shape: Shape{1, 96, 32, 32}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_472_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(768, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 32;
        const int in_width = 32;
        const int in_depth = 96;
        const int filter_height = 3;
        const int filter_width = 3;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 1;
        const int pad_width = 1;
        const int out_height = 32;
        const int out_width = 32;
        const int out_depth = 96;
        const int num_outputs = 98304;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
// Node name:	DepthwiseConv2dNative_478
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_468_0	type: float	shape: Shape{1, 96, 32, 32}
//	- name: Constant_423_0	type: float	shape: Shape{5, 5, 96, 1}
// Output:
//	- name: DepthwiseConv2dNative_478_0	type: float	shape: Shape{1, 96, 32, 32}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_478_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(768, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 32;
        const int in_width = 32;
        const int in_depth = 96;
        const int filter_height = 5;
        const int filter_width = 5;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 2;
        const int pad_width = 2;
        const int out_height = 32;
        const int out_width = 32;
        const int out_depth = 96;
        const int num_outputs = 98304;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Convolution_Convolution_DepthwiseConv2dNative_Convolution_DepthwiseConv2dNative_DepthwiseConv2dNative_Convolution_0(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* output0, float* output1, float* output2, float* output3, float* output4, float* output5, float* output6)
{
    __shared__ char shared_buffer[3072];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63)
    {
        Convolution_float_float_float_cuda_Convolution_471_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127)
    {
        Convolution_float_float_float_cuda_Convolution_471_block_kernel(input2, input3, output1, threadIdx.x, blockIdx.x - 64 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 895)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_472_block_kernel(input2, input4, output2, threadIdx.x, blockIdx.x - 128 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 896 && (int)blockIdx.x <= 959)
    {
        Convolution_float_float_float_cuda_Convolution_471_block_kernel(input2, input5, output3, threadIdx.x, blockIdx.x - 896 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 960 && (int)blockIdx.x <= 1727)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_478_block_kernel(input2, input6, output4, threadIdx.x, blockIdx.x - 960 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 1728 && (int)blockIdx.x <= 2495)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_472_block_kernel(input2, input7, output5, threadIdx.x, blockIdx.x - 1728 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 2496 && (int)blockIdx.x <= 2559)
    {
        Convolution_float_float_float_cuda_Convolution_471_block_kernel(input2, input8, output6, threadIdx.x, blockIdx.x - 2496 + 0, shared_buffer);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Convolution_Convolution_DepthwiseConv2dNative_Convolution_DepthwiseConv2dNative_DepthwiseConv2dNative_Convolution_0_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* input6, float* input7, float* input8, float* output0, float* output1, float* output2, float* output3, float* output4, float* output5, float* output6) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Convolution_Convolution_DepthwiseConv2dNative_Convolution_DepthwiseConv2dNative_DepthwiseConv2dNative_Convolution_0<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, input6, input7, input8, output0, output1, output2, output3, output4, output5, output6);
}
// Node name:	AvgPool_897
// Description:	AvgPool
// Input:
//	- name: Slice_895_0	type: float	shape: Shape{1, 64, 32, 32}
// Output:
//	- name: AvgPool_897_0	type: float	shape: Shape{1, 64, 16, 16}
void AvgPool_float_float_cuda_lib_AvgPool_897(hipdnnHandle_t cudnn_handle, float* input0, float* output0)
{
    hipdnnTensorDescriptor_t input_desc;
    CUDNN_SAFE_CALL(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_SAFE_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 64, 32, 32));
    hipdnnTensorDescriptor_t output_desc;
    CUDNN_SAFE_CALL(hipdnnCreateTensorDescriptor(&output_desc));
    CUDNN_SAFE_CALL(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 64, 16, 16));
    hipdnnPoolingDescriptor_t desc;
    hipdnnCreatePoolingDescriptor(&desc);
    CUDNN_SAFE_CALL(hipdnnSetPooling2dDescriptor(desc, HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING, HIPDNN_NOT_PROPAGATE_NAN,3, 3, 0, 0, 2, 2));
    const float alpha = 1.0;
    const float beta = 0.0;
    CUDNN_SAFE_CALL(hipdnnPoolingForward(cudnn_handle, desc, &alpha, input_desc, input0, &beta, output_desc, output0));
    CUDNN_SAFE_CALL(hipdnnDestroyTensorDescriptor(input_desc));
    CUDNN_SAFE_CALL(hipdnnDestroyTensorDescriptor(output_desc));
    CUDNN_SAFE_CALL(hipdnnDestroyPoolingDescriptor(desc));

}
// Node name:	 BlockFusion
// Input:
//	- name: Convolution_1529_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2953_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1462_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2744_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1531_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Slice_1484_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1535_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Add_1536_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// FusedKernel_float_float_float_float_cuda_Add_Add_59<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1529_0, Constant_2953_0, BatchNormInference_1462_0, Add_1535_0);
// FusedKernel_float_float_float_float_cuda_Add_Add_60<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1531_0, Constant_2744_0, Slice_1484_0, Add_1536_0);
// Deduped function map: <src_function_name : deduped_function_name>
// FusedKernel_float_float_float_float_cuda_Add_Add_60 : FusedKernel_float_float_float_float_cuda_Add_Add_59

// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_1529_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2953_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1462_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1535_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Add_float_float_float_cuda_Add_2610<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1529_0, Constant_2953_0, BatchNormInference_1533_0);
// Add_float_float_float_cuda_Add_1535<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(BatchNormInference_1533_0, BatchNormInference_1462_0, Add_1535_0);
__device__ __noinline__ void FusedKernel_float_float_float_float_cuda_Add_Add_59_block_kernel(float* input0, float* input1, float* input2, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(16, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    int tid = blockIdx.x * 512 + threadIdx.x;
    float temp0 = add(input0[tid], input1[tid]);
    float temp1 = add(temp0, input2[tid]);
    output0[tid] = temp1;

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_147(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* output0, float* output1)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15)
    {
        FusedKernel_float_float_float_float_cuda_Add_Add_59_block_kernel(input0, input1, input2, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31)
    {
        FusedKernel_float_float_float_float_cuda_Add_Add_59_block_kernel(input4, input3, input5, output1, threadIdx.x, blockIdx.x - 16 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_147_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_147<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, output0, output1);
}
