#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
// Node name:	Constant_2383
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2383_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2383(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2383_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2383_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[16384];
    bin_file.read(tmp_mem, 16384);
    hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_194
// Description:	Constant
// Input:
// Output:
//	- name: Constant_194_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_194(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_194_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_194_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4608];
    bin_file.read(tmp_mem, 4608);
    hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_33
// Description:	Constant
// Input:
// Output:
//	- name: Constant_33_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_33(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_33_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_33_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[2304];
    bin_file.read(tmp_mem, 2304);
    hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_251
// Description:	Constant
// Input:
// Output:
//	- name: Constant_251_0	type: float	shape: Shape{5, 5, 32, 1}
void Constant_float_cuda_Constant_251(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_251_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_251_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[3200];
    bin_file.read(tmp_mem, 3200);
    hipMemcpyAsync(output0, tmp_mem, 3200, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2689
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2689_0	type: float	shape: Shape{128, 768, 1, 1}
void Constant_float_cuda_Constant_2689(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2689_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2689_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[393216];
    bin_file.read(tmp_mem, 393216);
    hipMemcpyAsync(output0, tmp_mem, 393216, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2551
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2551_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2551(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2551_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2551_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2922
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2922_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2922(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2922_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2922_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_332
// Description:	Constant
// Input:
// Output:
//	- name: Constant_332_0	type: float	shape: Shape{5, 5, 32, 1}
void Constant_float_cuda_Constant_332(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_332_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_332_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[3200];
    bin_file.read(tmp_mem, 3200);
    hipMemcpyAsync(output0, tmp_mem, 3200, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2964
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2964_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2964(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2964_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2964_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_322
// Description:	Constant
// Input:
// Output:
//	- name: Constant_322_0	type: float	shape: Shape{5, 5, 32, 1}
void Constant_float_cuda_Constant_322(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_322_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_322_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[3200];
    bin_file.read(tmp_mem, 3200);
    hipMemcpyAsync(output0, tmp_mem, 3200, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2020
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2020_0	type: float	shape: Shape{32, 96, 1, 1}
void Constant_float_cuda_Constant_2020(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2020_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2020_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[12288];
    bin_file.read(tmp_mem, 12288);
    hipMemcpyAsync(output0, tmp_mem, 12288, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_3038
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3038_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_3038(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_3038_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_3038_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_30
// Description:	Constant
// Input:
// Output:
//	- name: Constant_30_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_30(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_30_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_30_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1152];
    bin_file.read(tmp_mem, 1152);
    hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	 BlockFusion
// Input:
//	- name: BatchNormInference_579_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Relu_580_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_49_0	type: float	shape: Shape{5, 5, 32, 1}
//	- name: Constant_1_0	type: float	shape: Shape{3, 3, 32, 1}
//	- name: Constant_57_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: Slice_582_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_585_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_583_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_584_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Slice_float_float_cuda_Slice_582<<<dim3(512, 1, 1), dim3(64, 1, 1), 0, 0>>>(BatchNormInference_579_0, Slice_582_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_585<<<dim3(256, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_580_0, Constant_49_0, DepthwiseConv2dNative_585_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_583<<<dim3(256, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_580_0, Constant_1_0, DepthwiseConv2dNative_583_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_584<<<dim3(256, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_580_0, Constant_57_0, DepthwiseConv2dNative_584_0);
// Deduped function map: <src_function_name : deduped_function_name>
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_584 : DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_583

// Node name:	Slice_582
// Description:	Slice
// Input:
//	- name: BatchNormInference_579_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Slice_582_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void Slice_float_float_cuda_Slice_582_block_kernel(float* input0, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 64){
        return;
    }
    const dim3 blockDim(64, 1, 1);
    const dim3 gridDim(512, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < 32768)
    {
        uint32_t input_strides[] = {32768, 1024, 32, 1};
        uint32_t output_strides[] = {32768, 1024, 32, 1};
        uint32_t lower_bounds[] = {0, 0, 0, 0};
        uint32_t slice_strides[] = {1, 1, 1, 1};
        uint32_t input_idx = 0;
        uint32_t output_idx = tid;
        input_idx += (((output_idx / output_strides[0]) * slice_strides[0]) + lower_bounds[0]) * input_strides[0];
        output_idx %= output_strides[0];
        input_idx += (((output_idx / output_strides[1]) * slice_strides[1]) + lower_bounds[1]) * input_strides[1];
        output_idx %= output_strides[1];
        input_idx += (((output_idx / output_strides[2]) * slice_strides[2]) + lower_bounds[2]) * input_strides[2];
        output_idx %= output_strides[2];
        input_idx += (((output_idx / output_strides[3]) * slice_strides[3]) + lower_bounds[3]) * input_strides[3];
        output0[tid] = input0[input_idx];
    }

}
// Node name:	DepthwiseConv2dNative_585
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_580_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_49_0	type: float	shape: Shape{5, 5, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_585_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_585_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(256, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 32;
        const int in_width = 32;
        const int in_depth = 32;
        const int filter_height = 5;
        const int filter_width = 5;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 2;
        const int pad_width = 2;
        const int out_height = 32;
        const int out_width = 32;
        const int out_depth = 32;
        const int num_outputs = 32768;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
// Node name:	DepthwiseConv2dNative_583
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_580_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_1_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_583_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_583_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(256, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 32;
        const int in_width = 32;
        const int in_depth = 32;
        const int filter_height = 3;
        const int filter_width = 3;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 1;
        const int pad_width = 1;
        const int out_height = 32;
        const int out_width = 32;
        const int out_depth = 32;
        const int num_outputs = 32768;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_11(float* input0, float* input1, float* input2, float* input3, float* input4, float* output0, float* output1, float* output2, float* output3)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 511)
    {
        Slice_float_float_cuda_Slice_582_block_kernel(input0, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 512 && (int)blockIdx.x <= 767)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_585_block_kernel(input1, input2, output1, threadIdx.x, blockIdx.x - 512 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 768 && (int)blockIdx.x <= 1023)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_583_block_kernel(input1, input3, output2, threadIdx.x, blockIdx.x - 768 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 1024 && (int)blockIdx.x <= 1279)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_583_block_kernel(input1, input4, output3, threadIdx.x, blockIdx.x - 1024 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_11_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* output0, float* output1, float* output2, float* output3) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_11<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, output0, output1, output2, output3);
}
// Node name:	 BlockFusion
// Input:
//	- name: BatchNormInference_767_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Relu_771_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_213_0	type: float	shape: Shape{5, 5, 32, 1}
//	- name: Constant_5_0	type: float	shape: Shape{3, 3, 32, 1}
//	- name: Constant_371_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: Slice_769_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_775_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_777_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_776_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Slice_float_float_cuda_Slice_769<<<dim3(512, 1, 1), dim3(64, 1, 1), 0, 0>>>(BatchNormInference_767_0, Slice_769_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_775<<<dim3(256, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_771_0, Constant_213_0, DepthwiseConv2dNative_775_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_777<<<dim3(256, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_771_0, Constant_5_0, DepthwiseConv2dNative_777_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_776<<<dim3(256, 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_771_0, Constant_371_0, DepthwiseConv2dNative_776_0);
// Deduped function map: <src_function_name : deduped_function_name>
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_776 : DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_777

// Node name:	Slice_769
// Description:	Slice
// Input:
//	- name: BatchNormInference_767_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Slice_769_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void Slice_float_float_cuda_Slice_769_block_kernel(float* input0, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 64){
        return;
    }
    const dim3 blockDim(64, 1, 1);
    const dim3 gridDim(512, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < 32768)
    {
        uint32_t input_strides[] = {32768, 1024, 32, 1};
        uint32_t output_strides[] = {32768, 1024, 32, 1};
        uint32_t lower_bounds[] = {0, 0, 0, 0};
        uint32_t slice_strides[] = {1, 1, 1, 1};
        uint32_t input_idx = 0;
        uint32_t output_idx = tid;
        input_idx += (((output_idx / output_strides[0]) * slice_strides[0]) + lower_bounds[0]) * input_strides[0];
        output_idx %= output_strides[0];
        input_idx += (((output_idx / output_strides[1]) * slice_strides[1]) + lower_bounds[1]) * input_strides[1];
        output_idx %= output_strides[1];
        input_idx += (((output_idx / output_strides[2]) * slice_strides[2]) + lower_bounds[2]) * input_strides[2];
        output_idx %= output_strides[2];
        input_idx += (((output_idx / output_strides[3]) * slice_strides[3]) + lower_bounds[3]) * input_strides[3];
        output0[tid] = input0[input_idx];
    }

}
// Node name:	DepthwiseConv2dNative_775
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_771_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_213_0	type: float	shape: Shape{5, 5, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_775_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_775_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(256, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 32;
        const int in_width = 32;
        const int in_depth = 32;
        const int filter_height = 5;
        const int filter_width = 5;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 2;
        const int pad_width = 2;
        const int out_height = 32;
        const int out_width = 32;
        const int out_depth = 32;
        const int num_outputs = 32768;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
// Node name:	DepthwiseConv2dNative_777
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_771_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_5_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_777_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __noinline__ void DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_777_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 128){
        return;
    }
    const dim3 blockDim(128, 1, 1);
    const dim3 gridDim(256, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);

        typedef float S;
        float *input = input0;
        float *filter = input1;
        float *output = output0;

        const int in_height = 32;
        const int in_width = 32;
        const int in_depth = 32;
        const int filter_height = 3;
        const int filter_width = 3;
        const int depth_multiplier = 1;
        const int stride = 1;
        const int pad_height = 1;
        const int pad_width = 1;
        const int out_height = 32;
        const int out_width = 32;
        const int out_depth = 32;
        const int num_outputs = 32768;

        for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < num_outputs;
             thread_id += blockDim.x * gridDim.x)
        {
            // Compute the indexes of this thread in the output.
            //
            // We want coalesced reads so we make sure that each warp reads
            // a contiguous chunk of memory.
            //
            // THIS IS PROBABLY WRONG, we are not doing coalesced reads
            // into the input, because of the depth multiplier division...
            const int out_col = thread_id % out_width;
            const int out_row = (thread_id / out_width) % out_height;
            const int out_channel = (thread_id / out_width / out_height) % out_depth;
            const int batch = thread_id / out_width / out_height / out_depth;

            // Compute the input depth and the index of depth multiplier
            // based off the output depth index that this thread is
            // computing n.
            const int in_channel = out_channel / depth_multiplier;
            const int multiplier = out_channel % depth_multiplier;

            // Data is stored in the following format (let's assume we
            // flatten the height and width into one contiguous dimension
            // called "P".
            //
            // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
            //
            // Each row contains in_depth * in_height * in_width values
            // for each sample in the batch.
            //
            // We can further flatten it into:
            //
            // B1C1P1 B1C1P2 .....
            // B1C2P1 B1C2P2 ....
            // B2C1P1 B2C1P2 .....
            // B2C2P1 B2C2P2 ....
            //
            // where each row is a contiguous array of all of the spatial
            // pixels for a given batch and input depth.  The following
            // loop #pragma unrolls across the filter dimensions for a given thread,
            // indexing into the filter value and the corresponding input
            // patch.
            //
            // We can compute the index into the patch once right here.
            const int input_offset_temp = (batch * in_depth + in_channel) * (in_height * in_width);

            // Finally, we can iterate over the spatial dimensions and perform the
            // convolution, writing into the output at the end.
            //
            // We perform an additional optimization, where we can determine
            // whether the patch fits within the image indices statically, and
            // avoid boundary checking within the loop.
            const int input_row_start = out_row * stride - pad_height;
            const int input_col_start = out_col * stride - pad_width;
            const int input_row_end = input_row_start + filter_height;
            const int input_col_end = input_col_start + filter_width;

            S sum = static_cast<S>(0);
            if (input_row_start >= 0 && input_col_start >= 0 && input_row_end < in_height &&
                input_col_end < in_width)
            {
                // Loop that doesn't need to check for boundary conditions.
                #pragma unroll
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;

                        const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;
                        const int filter_offset =
                            multiplier +
                            depth_multiplier *
                                (in_channel + in_depth * (filter_col + filter_offset_temp));
                        sum += static_cast<S>(__ldg(input + input_offset)) *
                               static_cast<S>(__ldg(filter + filter_offset));
                    }
                }
            }
            else
            {
                // Loop that needs to check for boundary conditions.
                #pragma unroll 
                for (int filter_row = 0; filter_row < filter_height; ++filter_row)
                {
                    const int in_row = input_row_start + filter_row;
                    const int filter_offset_temp = filter_width * filter_row;
                    #pragma unroll 
                    for (int filter_col = 0; filter_col < filter_width; ++filter_col)
                    {
                        const int in_col = input_col_start + filter_col;
                        // TODO(vrv): the in_row check can be done outside of this loop;
                        // benchmark both methods to determine the better decision.
                        if (in_row >= 0 && in_row < in_height && in_col >= 0 && in_col < in_width)
                        {
                            const int in_col = input_col_start + filter_col;

                            // input_offset_temp indexes into the start of memory
                            // where the spatial data starts.
                            const int input_offset = (input_offset_temp) + (in_row * in_width) + in_col;

                            const int filter_offset =
                                multiplier +
                                depth_multiplier *
                                    (in_channel + in_depth * (filter_col + filter_offset_temp));
                            sum += static_cast<S>(__ldg(input + input_offset)) *
                                   static_cast<S>(__ldg(filter + filter_offset));
                        }
                    }
                }
            }

            output[thread_id] = static_cast<S>(sum);
        }
        
}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_38(float* input0, float* input1, float* input2, float* input3, float* input4, float* output0, float* output1, float* output2, float* output3)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 511)
    {
        Slice_float_float_cuda_Slice_769_block_kernel(input0, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 512 && (int)blockIdx.x <= 767)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_775_block_kernel(input1, input2, output1, threadIdx.x, blockIdx.x - 512 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 768 && (int)blockIdx.x <= 1023)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_777_block_kernel(input1, input3, output2, threadIdx.x, blockIdx.x - 768 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 1024 && (int)blockIdx.x <= 1279)
    {
        DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_777_block_kernel(input1, input4, output3, threadIdx.x, blockIdx.x - 1024 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_38_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* output0, float* output1, float* output2, float* output3) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_38<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, output0, output1, output2, output3);
}
