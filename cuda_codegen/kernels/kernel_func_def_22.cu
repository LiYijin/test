#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1)
{
    return x0 + x1;
}
// Node name:	Constant_426
// Description:	Constant
// Input:
// Output:
//	- name: Constant_426_0	type: float	shape: Shape{5, 5, 128, 1}
void Constant_float_cuda_Constant_426(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_426_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_426_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[12800];
    bin_file.read(tmp_mem, 12800);
    hipMemcpyAsync(output0, tmp_mem, 12800, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_116
// Description:	Constant
// Input:
// Output:
//	- name: Constant_116_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_116(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_116_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_116_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[2304];
    bin_file.read(tmp_mem, 2304);
    hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2521
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2521_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2521(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2521_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2521_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2071
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2071_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2071(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2071_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2071_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4096];
    bin_file.read(tmp_mem, 4096);
    hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_57
// Description:	Constant
// Input:
// Output:
//	- name: Constant_57_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_57(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_57_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_57_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[1152];
    bin_file.read(tmp_mem, 1152);
    hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_250
// Description:	Constant
// Input:
// Output:
//	- name: Constant_250_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_250(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_250_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_250_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4608];
    bin_file.read(tmp_mem, 4608);
    hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2910
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2910_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2910(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2910_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2910_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2803
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2803_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2803(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2803_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2803_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_3174
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3174_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_3174(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_3174_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_3174_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[32768];
    bin_file.read(tmp_mem, 32768);
    hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2704
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2704_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2704(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2704_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2704_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2137
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2137_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2137(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2137_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2137_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[4096];
    bin_file.read(tmp_mem, 4096);
    hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	Constant_2530
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2530_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2530(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_2530_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_2530_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[65536];
    bin_file.read(tmp_mem, 65536);
    hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	 BlockFusion
// Input:
//	- name: Constant_2876_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1715_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Slice_1671_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1717_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2973_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1651_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1724_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Add_1725_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// FusedKernel_float_float_float_float_cuda_Add_Add_71<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1715_0, Constant_2876_0, Slice_1671_0, Add_1724_0);
// FusedKernel_float_float_float_float_cuda_Add_Add_72<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1717_0, Constant_2973_0, BatchNormInference_1651_0, Add_1725_0);
// Deduped function map: <src_function_name : deduped_function_name>
// FusedKernel_float_float_float_float_cuda_Add_Add_72 : FusedKernel_float_float_float_float_cuda_Add_Add_71

// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_1715_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2876_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Slice_1671_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1724_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Add_float_float_float_cuda_Add_2718<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1715_0, Constant_2876_0, BatchNormInference_1721_0);
// Add_float_float_float_cuda_Add_1724<<<dim3(16, 1, 1), dim3(512, 1, 1), 0, 0>>>(BatchNormInference_1721_0, Slice_1671_0, Add_1724_0);
__device__ __noinline__ void FusedKernel_float_float_float_float_cuda_Add_Add_71_block_kernel(float* input0, float* input1, float* input2, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 512){
        return;
    }
    const dim3 blockDim(512, 1, 1);
    const dim3 gridDim(16, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    int tid = blockIdx.x * 512 + threadIdx.x;
    float temp0 = add(input0[tid], input1[tid]);
    float temp1 = add(temp0, input2[tid]);
    output0[tid] = temp1;

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_174(float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* output0, float* output1)
{

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15)
    {
        FusedKernel_float_float_float_float_cuda_Add_Add_71_block_kernel(input1, input0, input2, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
    }
    else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31)
    {
        FusedKernel_float_float_float_float_cuda_Add_Add_71_block_kernel(input3, input4, input5, output1, threadIdx.x, blockIdx.x - 16 + 0, NULL);
    }

}
extern void BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_174_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* input3, float* input4, float* input5, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_174<<<grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4, input5, output0, output1);
}
// Node name:	Constant_290
// Description:	Constant
// Input:
// Output:
//	- name: Constant_290_0	type: float	shape: Shape{5, 5, 128, 1}
void Constant_float_cuda_Constant_290(hipStream_t stream, float* output0)
{
    std::ifstream bin_file("./Constant/Constant_290_0.bin" , std::ios::in | std::ios::binary);
    if(bin_file.fail())
    {
    	printf("Load Constant_290_0 failed.\n");
    	exit(1);
    }
    char* tmp_mem = new char[12800];
    bin_file.read(tmp_mem, 12800);
    hipMemcpyAsync(output0, tmp_mem, 12800, hipMemcpyHostToDevice, stream);
    bin_file.close();

}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_1336_0	type: float	shape: Shape{1, 384, 16, 16}
//	- name: Constant_1338_0	type: float	shape: Shape{}
//	- name: Constant_2509_0	type: float	shape: Shape{128, 384, 1, 1}
// Output:
//	- name: Pad_1339_0	type: float	shape: Shape{1, 384, 17, 17}
//	- name: Convolution_1341_0	type: float	shape: Shape{1, 128, 16, 16}
// Fused functions:
// Pad_float_float_float_cuda_Pad_1339<<<dim3(1734, 1, 1), dim3(64, 1, 1), 0, 0>>>(Relu_1336_0, Constant_1338_0, Pad_1339_0);
// Convolution_float_float_float_cuda_Convolution_1341<<<dim3(2, 8, 4), dim3(8, 2, 16), 0, 0>>>(Relu_1336_0, Constant_2509_0, Convolution_1341_0);
// Deduped function map: <src_function_name : deduped_function_name>

// Node name:	Pad_1339
// Description:	Pad
// Input:
//	- name: Relu_1336_0	type: float	shape: Shape{1, 384, 16, 16}
//	- name: Constant_1338_0	type: float	shape: Shape{}
// Output:
//	- name: Pad_1339_0	type: float	shape: Shape{1, 384, 17, 17}
__device__ __noinline__ void Pad_float_float_float_cuda_Pad_1339_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 64){
        return;
    }
    const dim3 blockDim(64, 1, 1);
    const dim3 gridDim(1734, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    float* in = input0;
    float* pad = input1;
    float* out = output0;
    if (tid < 110976)
    {
        size_t input_shape0 = 1;
        size_t input_shape1 = 384;
        size_t input_shape2 = 16;
        size_t input_shape3 = 16;
        uint32_t input_strides0 = 98304;
        uint32_t input_strides1 = 256;
        uint32_t input_strides2 = 16;
        uint32_t input_strides3 = 1;
        uint32_t output_strides0 = 110976;
        uint32_t output_strides1 = 289;
        uint32_t output_strides2 = 17;
        uint32_t output_strides3 = 1;
        uint32_t padding_below0 = 0;
        uint32_t padding_below1 = 0;
        uint32_t padding_below2 = 0;
        uint32_t padding_below3 = 0;
        uint32_t padding_interior0 = 0;
        uint32_t padding_interior1 = 0;
        uint32_t padding_interior2 = 0;
        uint32_t padding_interior3 = 0;
        bool in_bounds = true;
        uint32_t output_pixel = tid;
        uint32_t input_pixel = 0;
        int32_t input, input_dil;
        input_dil = output_pixel / output_strides0 - padding_below0;
        input = input_dil / (padding_interior0 + 1);
        input_dil %= (padding_interior0 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape0) && (input_dil == 0);
        input_pixel += input * input_strides0;
        output_pixel %= output_strides0;
        input_dil = output_pixel / output_strides1 - padding_below1;
        input = input_dil / (padding_interior1 + 1);
        input_dil %= (padding_interior1 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape1) && (input_dil == 0);
        input_pixel += input * input_strides1;
        output_pixel %= output_strides1;
        input_dil = output_pixel / output_strides2 - padding_below2;
        input = input_dil / (padding_interior2 + 1);
        input_dil %= (padding_interior2 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape2) && (input_dil == 0);
        input_pixel += input * input_strides2;
        output_pixel %= output_strides2;
        input_dil = output_pixel / output_strides3 - padding_below3;
        input = input_dil / (padding_interior3 + 1);
        input_dil %= (padding_interior3 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape3) && (input_dil == 0);
        input_pixel += input * input_strides3;
        out[tid] = (in_bounds) ? in[input_pixel] : *pad;
    }

}
// Node name:	Convolution_1341
// Description:	Convolution
// Input:
//	- name: Relu_1336_0	type: float	shape: Shape{1, 384, 16, 16}
//	- name: Constant_2509_0	type: float	shape: Shape{128, 384, 1, 1}
// Output:
//	- name: Convolution_1341_0	type: float	shape: Shape{1, 128, 16, 16}
__device__ __forceinline__ void Convolution_float_float_float_cuda_Convolution_1341_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    
    const dim3 blockDim(8, 2, 16);
    const dim3 gridDim(2, 8, 4);
    const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
    const dim3 blockIdx(block_id % 2, block_id / 2 % 8, block_id / 16);
    /*
    __shared__ float pad_temp_shared[512];
    __shared__ float input1_shared[1024];
    */
    //__shared__ char shared_buffer1[6144];
    float* pad_temp_shared = (float*)(shared_buffer + 0);
    float* input1_shared = (float*)(shared_buffer + 2048);
    
    {
        float* compute = output0;{
           float compute_local[2];
          
          
          compute_local[0] = 0.000000e+00f;
          compute_local[1] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2))];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7))];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4))];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 1)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 2)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 3)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 8192)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 8192)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 32)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 33)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 34)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 35)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 16384)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 16384)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 64)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 65)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 66)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 67)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 24576)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 24576)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 96)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 97)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 98)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 99)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 32768)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 32768)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 128)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 129)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 130)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 131)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 40960)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 40960)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 160)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 161)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 162)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 163)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 49152)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 49152)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 192)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 193)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 194)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 195)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 57344)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 57344)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 224)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 225)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 226)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 227)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 65536)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 65536)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 256)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 257)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 258)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 259)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 73728)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 73728)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 288)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 289)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 290)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 291)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 81920)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 81920)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 320)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 321)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 322)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 323)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 90112)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 90112)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 352)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 353)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 354)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 355)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          compute[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))] = compute_local[0];
          compute[(((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 4096)] = compute_local[1];
        }


    }

}
extern "C" __global__  void BlockFusionKernel_float_float_float_float_float_cuda_Pad_Convolution_119(float* input0, float* input1, float* input2, float* output0, float* output1)
{
    __shared__ char shared_buffer[6144];

    if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 1733)
    {
        Pad_float_float_float_cuda_Pad_1339_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    }
    else if ((int)blockIdx.x >= 1734 && (int)blockIdx.x <= 1797)
    {
        Convolution_float_float_float_cuda_Convolution_1341_block_kernel(input0, input2, output1, threadIdx.x, blockIdx.x - 1734 + 0, shared_buffer);
    }

}
extern "C"  __global__  void pad_test(float* input0, float* input1, float* output0)
{
    /*
    __shared__ char shared_buffer[6144];
    Pad_float_float_float_cuda_Pad_1339_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
    */
   uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    float* in = input0;
    float* pad = input1;
    float* out = output0;
    if (tid < 110976)
    {
        size_t input_shape0 = 1;
        size_t input_shape1 = 384;
        size_t input_shape2 = 16;
        size_t input_shape3 = 16;
        uint32_t input_strides0 = 98304;
        uint32_t input_strides1 = 256;
        uint32_t input_strides2 = 16;
        uint32_t input_strides3 = 1;
        uint32_t output_strides0 = 110976;
        uint32_t output_strides1 = 289;
        uint32_t output_strides2 = 17;
        uint32_t output_strides3 = 1;
        uint32_t padding_below0 = 0;
        uint32_t padding_below1 = 0;
        uint32_t padding_below2 = 0;
        uint32_t padding_below3 = 0;
        uint32_t padding_interior0 = 0;
        uint32_t padding_interior1 = 0;
        uint32_t padding_interior2 = 0;
        uint32_t padding_interior3 = 0;
        bool in_bounds = true;
        uint32_t output_pixel = tid;
        uint32_t input_pixel = 0;
        int32_t input, input_dil;
        input_dil = output_pixel / output_strides0 - padding_below0;
        input = input_dil / (padding_interior0 + 1);
        input_dil %= (padding_interior0 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape0) && (input_dil == 0);
        input_pixel += input * input_strides0;
        output_pixel %= output_strides0;
        input_dil = output_pixel / output_strides1 - padding_below1;
        input = input_dil / (padding_interior1 + 1);
        input_dil %= (padding_interior1 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape1) && (input_dil == 0);
        input_pixel += input * input_strides1;
        output_pixel %= output_strides1;
        input_dil = output_pixel / output_strides2 - padding_below2;
        input = input_dil / (padding_interior2 + 1);
        input_dil %= (padding_interior2 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape2) && (input_dil == 0);
        input_pixel += input * input_strides2;
        output_pixel %= output_strides2;
        input_dil = output_pixel / output_strides3 - padding_below3;
        input = input_dil / (padding_interior3 + 1);
        input_dil %= (padding_interior3 + 1);
        in_bounds = in_bounds && (input >= 0) && (input < input_shape3) && (input_dil == 0);
        input_pixel += input * input_strides3;
        out[tid] = (in_bounds) ? in[input_pixel] : *pad;
    }
}
extern "C" __global__  void conv_test(float* input0, float* input1, float* output0)
{
    
    __shared__ char shared_buffer[6144];
    Convolution_float_float_float_cuda_Convolution_1341_block_kernel(input0, input1, output0, threadIdx.x, blockIdx.x, shared_buffer);
    
   /*
   __shared__ float pad_temp_shared[512];
   __shared__ float input1_shared[1024];
   {
        float* compute = output0;{
           float compute_local[2];
          
          
          compute_local[0] = 0.000000e+00f;
          compute_local[1] = 0.000000e+00f;
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2))];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7))];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4))];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 1)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 2)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 3)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 8192)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 8192)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 32)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 33)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 34)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 35)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 16384)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 16384)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 64)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 65)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 66)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 67)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 24576)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 24576)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 96)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 97)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 98)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 99)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 32768)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 32768)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 128)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 129)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 130)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 131)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 40960)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 40960)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 160)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 161)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 162)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 163)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 49152)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 49152)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 192)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 193)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 194)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 195)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 57344)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 57344)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 224)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 225)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 226)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 227)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 65536)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 65536)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 256)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 257)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 258)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 259)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 73728)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 73728)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 288)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 289)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 290)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 291)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 81920)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 81920)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 320)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 321)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 322)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 323)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          __syncthreads();
          pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2))] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) >> 2) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 90112)];
          pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1)] = input0[(((((((((int)threadIdx.z) * 512) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.y) * 32)) + ((((((int)threadIdx.x) * 2) + 1) >> 3) * 16)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)) + 90112)];
          input1_shared[(((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4))] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 352)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 1)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 353)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 2)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 354)];
          input1_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 4)) + 3)] = input1[(((((((int)blockIdx.z) * 12288) + (((int)threadIdx.z) * 768)) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 4)) + 355)];
          __syncthreads();
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[(((int)threadIdx.z) * 32)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] * input1_shared[((((int)threadIdx.z) * 32) + 512)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 1)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 16)] * input1_shared[((((int)threadIdx.z) * 32) + 513)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 2)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 32)] * input1_shared[((((int)threadIdx.z) * 32) + 514)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 3)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 48)] * input1_shared[((((int)threadIdx.z) * 32) + 515)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 4)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 64)] * input1_shared[((((int)threadIdx.z) * 32) + 516)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 5)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 80)] * input1_shared[((((int)threadIdx.z) * 32) + 517)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 6)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 96)] * input1_shared[((((int)threadIdx.z) * 32) + 518)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 7)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 112)] * input1_shared[((((int)threadIdx.z) * 32) + 519)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 8)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 128)] * input1_shared[((((int)threadIdx.z) * 32) + 520)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 9)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 144)] * input1_shared[((((int)threadIdx.z) * 32) + 521)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 10)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 160)] * input1_shared[((((int)threadIdx.z) * 32) + 522)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 11)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 176)] * input1_shared[((((int)threadIdx.z) * 32) + 523)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 12)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 192)] * input1_shared[((((int)threadIdx.z) * 32) + 524)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 13)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 208)] * input1_shared[((((int)threadIdx.z) * 32) + 525)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 14)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 224)] * input1_shared[((((int)threadIdx.z) * 32) + 526)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 15)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 240)] * input1_shared[((((int)threadIdx.z) * 32) + 527)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 16)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 256)] * input1_shared[((((int)threadIdx.z) * 32) + 528)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 17)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 272)] * input1_shared[((((int)threadIdx.z) * 32) + 529)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 18)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 288)] * input1_shared[((((int)threadIdx.z) * 32) + 530)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 19)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 304)] * input1_shared[((((int)threadIdx.z) * 32) + 531)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 20)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 320)] * input1_shared[((((int)threadIdx.z) * 32) + 532)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 21)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 336)] * input1_shared[((((int)threadIdx.z) * 32) + 533)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 22)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 352)] * input1_shared[((((int)threadIdx.z) * 32) + 534)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 23)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 368)] * input1_shared[((((int)threadIdx.z) * 32) + 535)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 24)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 384)] * input1_shared[((((int)threadIdx.z) * 32) + 536)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 25)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 400)] * input1_shared[((((int)threadIdx.z) * 32) + 537)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 26)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 416)] * input1_shared[((((int)threadIdx.z) * 32) + 538)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 27)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 432)] * input1_shared[((((int)threadIdx.z) * 32) + 539)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 28)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 448)] * input1_shared[((((int)threadIdx.z) * 32) + 540)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 29)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 464)] * input1_shared[((((int)threadIdx.z) * 32) + 541)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 30)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 480)] * input1_shared[((((int)threadIdx.z) * 32) + 542)]));
          compute_local[0] = (compute_local[0] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 31)]));
          compute_local[1] = (compute_local[1] + (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) + 496)] * input1_shared[((((int)threadIdx.z) * 32) + 543)]));
          compute[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x))] = compute_local[0];
          compute[(((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.y) * 32)) + (((int)threadIdx.y) * 16)) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)) + 4096)] = compute_local[1];
        }
        

    }
    */
}



extern void BlockFusionKernel_float_float_float_float_float_cuda_Pad_Convolution_119_Call(const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream, float* input0, float* input1, float* input2, float* output0, float* output1) {
    BlockFusionKernel_float_float_float_float_float_cuda_Pad_Convolution_119<<<grids, blocks, mem, stream>>>(input0, input1, input2, output0, output1);
}
