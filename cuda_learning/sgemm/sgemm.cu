#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include <hipblas.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

//A: M*K
//B: K*N
//C: M*N
#define BLOCKDIM 16

void cpu_sgemm(const float *A, const float *B, float *C, int M, int N, int K)
{
  double tStart = cpuSecond();
  for(int i=0;i<M;i++){
    for(int j=0;j<N;j++)
      C[i*N+j]=0;
    for(int k=0;k<K;k++){
      for(int j=0;j<N;j++){
        C[i*N+j] += A[i*K+k] * B[k*N+j];
      }
    }
  }
  double tLast = cpuSecond()-tStart;
  printf("cpu:%.6f\n",tLast*1000.0);
}

__global__ void sgemm_v0(const float *A, const float *B, float *C, int M, int N, int K)
{
  int j = blockIdx.x*blockDim.x + threadIdx.x;
  int i = blockIdx.y*blockDim.y + threadIdx.y;
  if(i<M && j<N){
    float sum = 0;
    for(int k=0;k<K;k++)
      sum += A[i*K+k] * B[k*N+j];
    C[i*N+j] = sum;
  }
}

__global__ void sgemm_v1(const float *A, const float *B, float *C, int M, int N, int K)
{
  __shared__ float mm1[BLOCKDIM][BLOCKDIM];
  __shared__ float mm2[BLOCKDIM][BLOCKDIM];
  float sum=0;

  #pragma unroll 4
  for(int tileidx = 0;tileidx<K;tileidx += BLOCKDIM){
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int ix = blockIdx.x*blockDim.x+threadIdx.x;
    int j = tileidx + threadIdx.x;
    if(iy<M && j<K)
      mm1[threadIdx.y][threadIdx.x] = A[iy*K+j];
    if(j<K && ix<N)
      mm2[threadIdx.y][threadIdx.x] = B[j*N+ix];
    __syncthreads();
    #pragma unroll 4
    for(int k=0;k<BLOCKDIM && k+tileidx<K;k++)
      sum += mm1[threadIdx.y][k] * mm2[k][threadIdx.x];
    __syncthreads();
  }
  int i = blockIdx.y*blockDim.y+threadIdx.y;
  int j = blockIdx.x*blockDim.x+threadIdx.x;
  if(i<M && j<N)
    C[(i)*N+(j)] = sum;
}

#define UNROLLSIZE 4
#define TILESIZE (BLOCKDIM*UNROLLSIZE)
__global__ void sgemm_v2(const float *A, const float *B, float *C, int M, int N, int K)
{
  __shared__ float mm1[TILESIZE][TILESIZE];
  __shared__ float mm2[TILESIZE][TILESIZE];

  float sum[UNROLLSIZE][UNROLLSIZE]={{0}};
  #pragma unroll 4
  for(int tileidx = 0;tileidx<K;tileidx += TILESIZE){
    #pragma unroll 4
    for(int i2=0;i2<UNROLLSIZE;i2++){
      #pragma unroll 4
      for(int i1=0;i1<UNROLLSIZE;i1++){
        int iy = (blockIdx.y * blockDim.y + threadIdx.y)*UNROLLSIZE+i1;
        int ix = (blockIdx.x * blockDim.x + threadIdx.x)*UNROLLSIZE+i1;
        int j = tileidx + threadIdx.x*UNROLLSIZE+i2;
        if(iy<M && j<K)
          mm1[threadIdx.y*UNROLLSIZE+i1][threadIdx.x*UNROLLSIZE+i2] = A[iy*K+j];
        if(j<K && ix<N)
          mm2[threadIdx.y*UNROLLSIZE+i1][threadIdx.x*UNROLLSIZE+i2] = B[j*N+ix];
      }
    }
    __syncthreads();
    #pragma unroll 4
    for(int k=0;k<TILESIZE && k+tileidx<K;k++)
      #pragma unroll 4
      for(int i2=0;i2<UNROLLSIZE;i2++)
        #pragma unroll 4
        for(int i1=0;i1<UNROLLSIZE;i1++)
          sum[i2][i1] += mm1[threadIdx.y*UNROLLSIZE+i1][k] * mm2[k][threadIdx.x*UNROLLSIZE+i2];
    __syncthreads();
  }
  #pragma unroll 4
  for(int i2=0;i2<UNROLLSIZE;i2++){
    #pragma unroll 4
    for(int i1=0;i1<UNROLLSIZE;i1++){
      int i = (blockIdx.y*blockDim.y+threadIdx.y)*UNROLLSIZE+i2;
      int j = (blockIdx.x*blockDim.x+threadIdx.x)*UNROLLSIZE+i1;
      if(i<M && j<N)
        C[(i)*N+(j)] = sum[i2][i1];
    }
  }
}

void gpu_sgemm(const float *A, const float *B, float *C, int M, int N, int K)
{
  dim3 block(BLOCKDIM,BLOCKDIM);
  dim3 grid((N-1)/block.x+1,(M-1)/block.y+1);
  double tStart,tLast;
  tStart = cpuSecond();
  sgemm_v0<<<grid,block>>>(A,B,C,M,N,K);
  hipDeviceSynchronize();
  tLast = cpuSecond()-tStart;
  printf("gpuv0:%.6f\n",tLast*1000.0);

  tStart = cpuSecond();
  sgemm_v2<<<grid,block>>>(A,B,C,M,N,K);
  hipDeviceSynchronize();
  tLast = cpuSecond()-tStart;
  printf("gpuv1:%.6f\n",tLast*1000.0);
}
void cublas_sgemm(const float *A, const float *B, float *C, int M, int N, int K)
{
  hipblasHandle_t handle;
  float alpha=1, beta=0;
  CHECK_CUBLAS(hipblasCreate(&handle));
  double tStart = cpuSecond();
  //(BtAt)t = AB
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
      N, M, K, 
      &alpha, 
      B, N,
      A, K, 
      &beta, 
      C, N);
  hipDeviceSynchronize();
  double tLast = cpuSecond()-tStart;
  printf("cublas:%.6f\n",tLast*1000.0);
  hipblasDestroy(handle);
}


int main(int argc,char **argv)
{
  float *A,*B,*C,*C_ref;
  int M=1000,N=1024,K=1700;
  if(argc==4){
    M = atoi(argv[1]);
    N = atoi(argv[2]);
    K = atoi(argv[3]);
  }
  A = (float*)malloc(sizeof(float)*M*K);
  B = (float*)malloc(sizeof(float)*K*N);
  C = (float*)malloc(sizeof(float)*M*N);
  C_ref = (float*)malloc(sizeof(float)*M*N);
  initialData(A,M*K);
  initialData(B,K*N);

  float *A_d, *B_d, *C_d;
  CHECK(hipMalloc((void**)&A_d,M*K*sizeof(float)));
  CHECK(hipMalloc((void**)&B_d,K*N*sizeof(float)));
  CHECK(hipMalloc((void**)&C_d,M*N*sizeof(float)));
  CHECK(hipMemcpy(A_d,A,M*K*sizeof(float),hipMemcpyHostToDevice));
  CHECK(hipMemcpy(B_d,B,K*N*sizeof(float),hipMemcpyHostToDevice));

  cpu_sgemm(A,B,C_ref,M,N,K);

  gpu_sgemm(A_d,B_d,C_d,M,N,K);
  CHECK(hipMemcpy(C,C_d,M*N*sizeof(float),hipMemcpyDeviceToHost));

  cublas_sgemm(A_d,B_d,C_d,M,N,K);
  //CHECK(hipMemcpy(C_ref,C_d,M*N*sizeof(float),hipMemcpyDeviceToHost));
  
  checkResult(C,C_ref,M*N);

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
  //printMatrix(C,M,N);
  //printMatrix(C_ref,M,N);
  return 0;
}