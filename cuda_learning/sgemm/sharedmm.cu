#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cuda_utils.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>
extern "C" __global__  void conv_test(float* input0, float* input1, float* output0);


int main()
{
    float *input0,*input1,*output;
    hipMalloc((void**)&input0,384*16*16*4);
    hipMalloc((void**)&input1,128*384*4);
    hipMalloc((void**)&output,128*16*16*4);
    for(int i=0;i<1000;i++){
    conv_test<<<dim3(64, 1, 1), dim3(256, 1, 1), 0, 0>>>(input0,input1,output);
    hipDeviceSynchronize();
    }
    double tStart = cpuSecond();
    for(int i=0;i<1000;i++){
    conv_test<<<dim3(64, 1, 1), dim3(256, 1, 1), 0, 0>>>(input0,input1,output);
    hipDeviceSynchronize();
    }
    double tLast = cpuSecond()-tStart;
    printf("time:%.6fms mean:%.6fus\n",tLast*1000.0,tLast*1000);
    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; // add
    hipProfilerStop();
  return 0;
}